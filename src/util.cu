#include "hip/hip_runtime.h"
#ifndef UTIL_HPP
#include "util.cuh"
#endif

void printTree(Node* node, int grpID)
{
    if (grpID == -1) {
        if (node->parent == nullptr)
            std::cout << std::setw(7) << node->identifier << ": " << node->branchLength << "\t" << "ROOT\t" << node->grpID << std::endl;
        else
            std::cout << std::setw(7) << node->identifier << ": " << node->branchLength << "\t" << node->parent->identifier << "\t" << node->grpID << std::endl;  

        if (node->children.size() == 0) return;
        // std::cout << "Print children\n";
        for (auto &c: node->children) printTree(c, -1);
    }
    else {
        if (node->grpID == grpID) {
            if (node->parent == nullptr)
                std::cout << std::setw(7) << node->identifier << ": " << node->branchLength << "\t" << "ROOT\t" << node->grpID << std::endl;
            else
                std::cout << std::setw(7) << node->identifier << ": " << node->branchLength << "\t" << node->parent->identifier << "\t" << node->grpID << std::endl;  
            if (node->children.size() == 0) return;
            // std::cout << "Print children\n";
        }
        for (auto &c: node->children) printTree(c, grpID);
    }
    
}

void printLeaves(Node* node)
{
    if (node->children.size() == 0) {
        std::cout << std::setw(7) << node->identifier << ": " << node->branchLength << "\t" << node->parent->identifier << '\t' << node->grpID << std::endl;
        return;
    }
    for (auto &c: node->children) printLeaves(c);
}

Tree* readNewick(po::variables_map& vm)
{
    auto treeBuiltStart = std::chrono::high_resolution_clock::now();
    std::string treeFileName = vm["tree"].as<std::string>();
    std::ifstream inputStream(treeFileName);
    if (!inputStream) { fprintf(stderr, "Error: Can't open file: %s\n", treeFileName.c_str()); exit(1); }
    std::string newick; inputStream >> newick;
    Tree *T = new Tree(newick);
    auto treeBuiltEnd = std::chrono::high_resolution_clock::now();
    std::chrono::nanoseconds treeBuiltTime = treeBuiltEnd - treeBuiltStart;
    std::cout << "Newick string read in: " <<  treeBuiltTime.count() << " ns\n";

    // printTree(T->root);

    return T;
}

Tree* readNewick(std::string treeFileName)
{
    auto treeBuiltStart = std::chrono::high_resolution_clock::now();
    std::ifstream inputStream(treeFileName);
    if (!inputStream) { fprintf(stderr, "Error: Can't open file: %s\n", treeFileName.c_str()); exit(1); }
    std::string newick; inputStream >> newick;
    Tree *T = new Tree(newick);
    auto treeBuiltEnd = std::chrono::high_resolution_clock::now();
    std::chrono::nanoseconds treeBuiltTime = treeBuiltEnd - treeBuiltStart;
    std::cout << "Newick string read in: " <<  treeBuiltTime.count() << " ns\n";

    return T;
}

void readFreq(std::string tempDir, Tree* tree, paritionInfo_t* partition, msa::utility* util) {
    for (auto subroot:  partition->partitionsRoot) {
        int subtree = tree->allNodes[subroot.first]->grpID;
        std::string freqFile = tempDir + '/' + "subtree-" + std::to_string(subtree) + ".freq.txt";
        std::ifstream inputStream(freqFile);
        if (!inputStream) { fprintf(stderr, "Error: Can't open file: %s\n", freqFile.c_str()); exit(1); }
        std::vector<std::vector<uint16_t>> freq;
        util->profileFreq[subtree] = freq;
        std::string rawInput;
        int idx; size_t seqNum, seqLen; 
        getline(inputStream, rawInput);
        std::string num = "";
        std::vector<uint16_t> numbers;
        for (int i = 0; i < rawInput.size(); ++i) {
            if (rawInput[i] == ',') {
                numbers.push_back(std::atoi(num.c_str()));
                num = "";
            }
            else if (i == rawInput.size()-1) {
                num += rawInput[i];
                numbers.push_back(std::atoi(num.c_str()));
                num = "";
            }
            else num += rawInput[i];
        }
        assert(numbers.size() == 3);
        idx = numbers[0];
        seqNum = numbers[1];
        seqLen = numbers[2]; 
        assert(idx == subtree);
        numbers.clear();
        util->seqsLen[subroot.first] = seqLen;
        for (int i = 0; i < 6; ++i) {
            std::vector<uint16_t> charFreq;
            util->profileFreq[subtree].push_back(charFreq);
            getline(inputStream, rawInput);
            for (int j = 0; j < rawInput.size(); ++j) {
                if (rawInput[j] == ',') {
                    util->profileFreq[subtree][i].push_back(std::atoi(num.c_str()));
                    num = "";
                }
                else if (j == rawInput.size()-1) {
                    num += rawInput[j];
                    util->profileFreq[subtree][i].push_back(std::atoi(num.c_str()));
                    num = "";
                }
                else num += rawInput[j];
            }
            if (util->profileFreq[subtree][i].size() > util->seqLen) util->seqLen = util->profileFreq[subtree][i].size();
        }
        inputStream.close();
        std::remove(freqFile.c_str());
    }
    return;
}

void getFreq(Tree* tree, paritionInfo_t* partition, msa::utility* util) {
    for (auto subroot:  partition->partitionsRoot) {
        int subtree = tree->allNodes[subroot.first]->grpID;
        size_t seqLen = util->seqsLen[subroot.first]; 
        std::vector<std::vector<uint16_t>> freq;
        util->profileFreq[subtree] = freq;
        for (int i = 0; i < 6; ++i) {
            std::vector<uint16_t> temp;
            util->profileFreq[subtree].push_back(temp);
            for (int j = 0; j < seqLen; ++j) {
                util->profileFreq[subtree][i].push_back(0);
            }
        }
        
        for (auto sIdx: tree->allNodes[subroot.first]->msaIdx) { 
            tbb::parallel_for(tbb::blocked_range<int>(0, seqLen), [&](tbb::blocked_range<int> r) {
            for (int s = r.begin(); s < r.end(); ++s) {
            // for (int s = 0; s < refLen; ++s) {
                if      (util->seqs[sIdx][s] == 'A' || util->seqs[sIdx][s] == 'a') util->profileFreq[subtree][0][s]+=1;
                else if (util->seqs[sIdx][s] == 'C' || util->seqs[sIdx][s] == 'c') util->profileFreq[subtree][1][s]+=1;
                else if (util->seqs[sIdx][s] == 'G' || util->seqs[sIdx][s] == 'g') util->profileFreq[subtree][2][s]+=1;
                else if (util->seqs[sIdx][s] == 'T' || util->seqs[sIdx][s] == 't' ||
                         util->seqs[sIdx][s] == 'U' || util->seqs[sIdx][s] == 'u') util->profileFreq[subtree][3][s]+=1;
                else if (util->seqs[sIdx][s] == 'N' || util->seqs[sIdx][s] == 'n') util->profileFreq[subtree][4][s]+=1;
                else                                                               util->profileFreq[subtree][5][s]+=1;
            }
            });
        }
    }
    // std::cout << "Finish getting frequency,\n";
    return;
}

void getPostOrderList(Node* node, std::stack<Node*>& postStack) {
    std::stack<Node*> s1;
    s1.push(node); 
    Node* current; 
  
    while (!s1.empty()) { 
        current = s1.top(); 
        postStack.push(current);
        s1.pop(); 
        for (auto ch: current->children) {
            if (ch->grpID == current->grpID) {
                s1.push(ch);
            }      
        }
    } 
    return;
}

bool cmp(std::string a, std::string b) {
    if (a.size() != b.size()) return a.size() < b.size();
    return a < b;
}

void outputFile(std::string fileName, msa::utility* util, Tree* T, int grpID) {
    std::ofstream outFile(fileName);
    if (!outFile) {
        fprintf(stderr, "ERROR: cant open file: %s\n", fileName.c_str());
        exit(1);
    }
    std::vector<std::string> seqs;
    for (auto seq: util->seqsIdx) {
        seqs.push_back(seq.first);
    }
    std::sort(seqs.begin(), seqs.end(), cmp);
    for (int s = 0; s < seqs.size(); ++s) {
        outFile << '>' << seqs[s] << "\n";
        int sIdx = util->seqsIdx[seqs[s]];
        int i = 0;
        while (util->seqs[sIdx][i] != 0) {
            outFile << util->seqs[sIdx][i];
            ++i;
        }
        // std::cout << seq.first << ':' << i << '\n'; 
        outFile << '\n';
    }
    outFile.close();
}

void outputFreq(std::string fileName, msa::utility* util, Tree* T, int grpID) {
    std::ofstream outFile(fileName);
    if (!outFile) {
        fprintf(stderr, "ERROR: cant open file: %s\n", fileName.c_str());
        exit(1);
    }
    // Info subtreeIdx, seqNum, seqLen
    outFile << grpID << ',' << util->seqNum << ',' << util->seqsLen[T->root->identifier] << '\n';

    size_t seqLen = util->seqsLen[T->root->identifier];
    std::cout << "seqLen: " << seqLen << '\n';
    uint16_t** freq = new uint16_t* [6];
    for (int i = 0; i < 6; ++i) {
        freq[i] = new uint16_t [seqLen];
        for (int j = 0; j <  seqLen; ++j) freq[i][j] = 0;
    }
    
    for (int sIdx = 0; sIdx < util->seqNum; ++sIdx) {
        for (int j = 0; j <  seqLen; ++j) {
            if      (util->seqs[sIdx][j] == 'A' || util->seqs[sIdx][j] == 'a') freq[0][j]+=1;
            else if (util->seqs[sIdx][j] == 'C' || util->seqs[sIdx][j] == 'c') freq[1][j]+=1;
            else if (util->seqs[sIdx][j] == 'G' || util->seqs[sIdx][j] == 'g') freq[2][j]+=1;
            else if (util->seqs[sIdx][j] == 'T' || util->seqs[sIdx][j] == 't' ||
                     util->seqs[sIdx][j] == 'U' || util->seqs[sIdx][j] == 'u') freq[3][j]+=1;
            else if (util->seqs[sIdx][j] == 'N' || util->seqs[sIdx][j] == 'n') freq[4][j]+=1;
            else                                                               freq[5][j]+=1;
        }
    }
    for (int i = 0; i < 6; ++i) {
        for (int j = 0; j < seqLen-1; ++j) {
            outFile << freq[i][j] << ',';
        }
        outFile << freq[i][seqLen-1] << '\n';
    }
    outFile.close();
    for (int i = 0; i < 6; ++i) delete [] freq[i];
    delete [] freq;
}

void outputSubtreeSeqs(std::string fileName, std::map<std::string, std::string> seqs) {
    std::ofstream outFile(fileName);
    if (!outFile) {
        fprintf(stderr, "ERROR: cant open file: %s\n", fileName.c_str());
        exit(1);
    }
    
    for (auto it = seqs.begin(); it != seqs.end(); ++it) {
        outFile << '>' << it->first << '\n';
        outFile << it->second << '\n';
    }

    outFile.close();
}

void getSubtreeNewick(Node* root, std::string& outputString) {
	if(root->children.size() != 0) {
		outputString += "(";
		for(int n = 0; n < root->children.size(); ++n) {
			if(n != 0) outputString += ",";
			getSubtreeNewick(root->children[n], outputString);
		}
		outputString += ")";
	}
	else {
		outputString += (root->identifier + ':' + std::to_string(root->branchLength));
    }
}

void outputSubtree(std::string fileName, Tree* T) {
	std::string out_str = "";
	getSubtreeNewick(T->root, out_str);
	out_str += ";\n";
	std::ofstream outFile(fileName);
    if (!outFile) {
        fprintf(stderr, "ERROR: cant open file: %s\n", fileName.c_str());
        exit(1);
    }
	outFile << out_str;
	outFile.close();
}

/*
void createOverlapMSA(Tree* tree, std::vector<std::pair<Node*, Node*>> nodes, msa::utility* util, Params& param)
{


    int numBlocks = 1024; 
    int blockSize = THREAD_NUM;

    // int alignSize = nodes.size() < numBlocks ? nodes.size() : numBlocks;
    // get maximum sequence/profile length 
    int32_t seqLen = util->memLen;
    int round = nodes.size() / numBlocks + 1;

    for (int r = 0; r < round; ++r) {
        int alignSize = (nodes.size() - r*numBlocks) < numBlocks ? (nodes.size() - r*numBlocks) : numBlocks;
        if (alignSize == 0) break;
        // store all sequences to array
        int32_t seqNum = 0;
        int32_t pairNum = alignSize;
        std::vector<std::string> seqs;
        // std::vector<std::vector<uint16_t>> freq;
        std::vector<uint16_t*> freq;
        std::vector<std::pair<int32_t, int32_t>> seqIdx;
        std::vector<std::pair<int32_t, int32_t>> len;
        // store info to array 
        auto freqStart = std::chrono::high_resolution_clock::now();
        for (int n = 0; n < alignSize; ++n) {
            int32_t nIdx = n + r*numBlocks;
            int32_t qryIdx = 0;
            int32_t refIdx = 0;
            int32_t qryLen = tree->allNodes[nodes[nIdx].second->identifier]->msa[0].size();
            int32_t refLen = tree->allNodes[nodes[nIdx].first->identifier]->msa[0].size();
            refIdx = seqNum;
            std::cout << n << "Len: " << refLen << ',' << qryLen << '\n';
            
            uint16_t *temp = new uint16_t[12*seqLen]; 
            for (int i = 0; i < 12*seqLen; ++i) temp[i]=0;
            // assert(temp.size() == 12*seqLen);
            // tbb::blocked_range<int> rangeRef(0, refLen);
            for (auto seq: tree->allNodes[nodes[nIdx].first->identifier]->msa) {
                tbb::parallel_for(tbb::blocked_range<int>(0, refLen), [&](tbb::blocked_range<int> r) {
                for (int s = r.begin(); s < r.end(); ++s) {
                    if      (seq[s] == 'A' || seq[s] == 'a')    temp[6*s+0]+=1;
                    else if (seq[s] == 'C' || seq[s] == 'c')    temp[6*s+1]+=1;
                    else if (seq[s] == 'G' || seq[s] == 'g')    temp[6*s+2]+=1;
                    else if (seq[s] == 'T' || seq[s] == 't' ||
                             seq[s] == 'U' || seq[s] == 'u')    temp[6*s+3]+=1;
                    else if (seq[s] == 'N' || seq[s] == 'n')    temp[6*s+4]+=1;
                    else                                        temp[6*s+5]+=1;
                }
                });
                ++seqNum;
                seqs.push_back(seq);
            }
            qryIdx = seqNum;
            for (auto seq: tree->allNodes[nodes[nIdx].second->identifier]->msa) {
                tbb::parallel_for(tbb::blocked_range<int>(0, qryLen), [&](tbb::blocked_range<int> r) {
                for (int s = r.begin(); s < r.end(); ++s) {
                    if      (seq[s] == 'A' || seq[s] == 'a') temp[6*(seqLen+s)+0]+=1;
                    else if (seq[s] == 'C' || seq[s] == 'c') temp[6*(seqLen+s)+1]+=1;
                    else if (seq[s] == 'G' || seq[s] == 'g') temp[6*(seqLen+s)+2]+=1;
                    else if (seq[s] == 'T' || seq[s] == 't'||
                             seq[s] == 'U' || seq[s] == 'u') temp[6*(seqLen+s)+3]+=1;
                    else if (seq[s] == 'N' || seq[s] == 'n') temp[6*(seqLen+s)+4]+=1;
                    else                                     temp[6*(seqLen+s)+5]+=1;
                }
                });
                ++seqNum;
                seqs.push_back(seq);
            }
            // printf("len: (%d, %d), num: (%d, %d)\n", refLen, qryLen, refIdx, qryIdx);
            seqIdx.push_back(std::make_pair(refIdx, qryIdx));
            len.push_back(std::make_pair(refLen, qryLen));
            freq.push_back(temp);
        }
        for (int l = 0; l < 100; l++) std::cout << freq[0][l] << ',';
                std::cout << '\n';
        auto freqEnd = std::chrono::high_resolution_clock::now();
        std::chrono::nanoseconds freqTime = freqEnd -freqStart;
        printf("Preprocessing time : %d ms\n",  (freqTime.count() / 1000000));
        // Malloc
        uint16_t* hostFreq = (uint16_t*)malloc(12*seqLen * pairNum * sizeof(uint16_t));
        int8_t* hostAln = (int8_t*)malloc(2*seqLen * pairNum * sizeof(int8_t));
        int32_t* hostLen = (int32_t*)malloc(2*pairNum * sizeof(int32_t));
        int32_t* hostAlnLen = (int32_t*)malloc(pairNum * sizeof(int32_t));
        int32_t* hostSeqInfo = (int32_t*)malloc(5 * sizeof(int32_t));
        paramType* hostParam = (paramType*)malloc(28 * sizeof(paramType)); 
        
        // Store Info to host mem
        for (int j = 0; j < 2*pairNum; ++j) { 
            if (j%2 == 0) hostLen[j] = len[j/2].first;
            else          hostLen[j] = len[j/2].second;
        }
        for (int j = 0; j < pairNum; ++j) {
            for (int l = 0; l < 12*seqLen; ++l) {
                hostFreq[12*seqLen*j+l] = freq[j][l];
            }
        }
        for (int j = 0; j < 2*seqLen*pairNum; ++j) { 
            hostAln[j] = 0;
        }
        for (int j = 0; j < pairNum; ++j) { 
            hostAlnLen[j] = 0;
        }
        hostSeqInfo[0] = seqLen;
        hostSeqInfo[1] = seqNum;
        hostSeqInfo[2] = pairNum;
        hostSeqInfo[3] = numBlocks;
        hostSeqInfo[4] = param.scoreMode;
       
        if (param.scoreMode == 0) {
            for (int i = 0; i < 5; ++i) {
                for (int j = 0; j < 5; ++j) {
                    if (i == 5 || j == 5)          hostParam[i*5+j] = 0;
                    else if (i == j)               hostParam[i*5+j] = param.match;
                    else if (i-j == 2 || j-i == 2) hostParam[i*5+j] = param.trans;
                    else                           hostParam[i*5+j] = param.mismatch;
                }
            }
            hostParam[25] = param.gapOpen;
            hostParam[26] = param.gapExtend;
            hostParam[27] = param.xdrop;
        }
        else if (param.scoreMode == 1) {
            for (int i = 0; i < 5; ++i) for (int j = 0; j < 5; ++j) hostParam[i*5+j] = param.hoxd70[i][j];
            hostParam[25] = param.hoxd70_gapOpen;
            hostParam[26] = param.hoxd70_gapExtend;
            hostParam[27] = param.xdrop;
        }

        // Cuda Malloc
        uint16_t* deviceFreq;
        int8_t* deviceAln;
        int32_t* deviceLen;
        int32_t* deviceAlnLen;
        int32_t* deviceSeqInfo;
        paramType* deviceParam;
        auto kernelStart = std::chrono::high_resolution_clock::now();
        hipMalloc((void**)&deviceFreq, 12*seqLen * pairNum * sizeof(uint16_t));
        hipMalloc((void**)&deviceAln, 2*seqLen * pairNum * sizeof(int8_t));
        hipMalloc((void**)&deviceLen, 2*pairNum * sizeof(int32_t));
        hipMalloc((void**)&deviceAlnLen, pairNum * sizeof(int32_t));
        hipMalloc((void**)&deviceSeqInfo, 5 * sizeof(int32_t));
        hipMalloc((void**)&deviceParam, 28 * sizeof(paramType));
        // Copy to device
        hipMemcpy(deviceFreq, hostFreq, 12*seqLen * pairNum * sizeof(uint16_t), hipMemcpyHostToDevice);
        hipMemcpy(deviceAln, hostAln, 2*seqLen * pairNum * sizeof(int8_t), hipMemcpyHostToDevice);
        hipMemcpy(deviceLen, hostLen, 2*pairNum * sizeof(int32_t), hipMemcpyHostToDevice);
        hipMemcpy(deviceAlnLen, hostAlnLen, pairNum * sizeof(int32_t), hipMemcpyHostToDevice);
        hipMemcpy(deviceSeqInfo, hostSeqInfo, 5 * sizeof(int32_t), hipMemcpyHostToDevice);
        hipMemcpy(deviceParam, hostParam, 28 * sizeof(paramType), hipMemcpyHostToDevice);

        // printf("Before kernel %s\n", hipGetErrorString(hipGetLastError()));
        alignGrpToGrp_talco<<<numBlocks, blockSize>>>(
            deviceFreq,
            deviceAln, 
            deviceLen,
            deviceAlnLen,
            deviceSeqInfo, 
            deviceParam
        );

        hipDeviceSynchronize();
        // printf("After kernel %s\n", hipGetErrorString(hipGetLastError()));
        // Copy to host
        hipMemcpy(hostAln, deviceAln, 2*seqLen * pairNum * sizeof(int8_t), hipMemcpyDeviceToHost);
        hipMemcpy(hostAlnLen, deviceAlnLen, pairNum * sizeof(int32_t), hipMemcpyDeviceToHost);
        auto kernelEnd = std::chrono::high_resolution_clock::now();
        std::chrono::nanoseconds kernelTime = kernelEnd - kernelStart;
        for (int n = 0; n <  pairNum; ++n) {
            std::cout << n << ':' << hostAlnLen[n] << '\n';
            // if (hostAlnLen[gn][n] > maxAlnLen) maxAlnLen = hostAlnLen[gn][n];
        }
        if (round > 1) {
            printf("Round. %d align %d pairs. KernelTime: %d ms\n", r, alignSize, kernelTime.count() / 1000000);
        }
        else {
            std::cout << "GPU KernelTime "<< kernelTime.count() / 1000000<< " ms\n";
        }
        auto reAlnStart = std::chrono::high_resolution_clock::now();
        for (int k = 0; k < pairNum; ++k) {
            // std::vector<std::string> alignment;
            int32_t refNum = seqIdx[k].second - seqIdx[k].first;
            int32_t qryNum = (k != pairNum-1) ? seqIdx[k+1].first - seqIdx[k].second : seqNum - seqIdx[k].second;
            int32_t refStart = seqIdx[k].first;
            int32_t qryStart = seqIdx[k].second;
            int32_t refIndex = 0;
            int32_t qryIndex = 0;
            // printf("k: %d, refNum: %d, qryNum: %d\n", k, refNum, qryNum);
            // printf("k: %d, length: %d\n", k, hostAlnLen[k]);
            // for (int j = 0; j < qryNum + refNum; ++j) alignment.push_back("");
            int nIdx = k + r*numBlocks;
            // printf("k: %d, length: %d, %s\n", k, hostAlnLen[k], nodes[nIdx].first->identifier.c_str());
            if (hostAlnLen[k] <= 0) {
                int32_t refLen = util->seqsLen[nodes[nIdx].first->identifier];
                int32_t qryLen = util->seqsLen[nodes[nIdx].second->identifier];
                std::vector<std::string> reference, query;
                std::vector<int8_t> aln;
                for (auto s: tree->allNodes[nodes[nIdx].first->identifier]->msa) reference.push_back(s);
                for (auto s: tree->allNodes[nodes[nIdx].second->identifier]->msa) query.push_back(s);
                alignGrpToGrp_traditional
                (
                    freq[k],
                    seqLen,
                    refLen,
                    qryLen,
                    param,
                    aln
                );
                int32_t alnLen = aln.size();
                util->memCheck(alnLen);
                std::reverse(aln.begin(), aln.end());
                // for (int j = 0; j < aln.size(); ++j) {
                //     // std::cout << j << ',' << refIndex << ',' << qryIndex << '\n';
                //     if ((aln[j] & 0xFFFF) == 0) {
                //         for (size_t i=0; i<refNum; i++) alignment[i]        += reference[i][refIndex]; 
                //         for (size_t i=0; i<qryNum; i++) alignment[i+refNum] += query[i][qryIndex];
                //         qryIndex++;refIndex++;
                //     }
                //     else if ((aln[j] & 0xFFFF) == 2) {
                //         for (size_t i=0; i<refNum; i++) alignment[i]        += reference[i][refIndex]; 
                //         for (size_t i=0; i<qryNum; i++) alignment[i+refNum] += '-';
                //             refIndex++;
                //         }
                //     else {
                //         for (size_t i=0; i<refNum; i++) alignment[i]        += '-'; 
                //         for (size_t i=0; i<qryNum; i++) alignment[i+refNum] += query[i][qryIndex];
                //         qryIndex++;
                //     }
                // }
                tree->allNodes[nodes[nIdx].first->identifier]->msaAln = aln;
                printf("CPU fallback (traditional global alignment) on No. %d (%s), Alignment Length: %d\n", k, tree->allNodes[nodes[nIdx].first->identifier]->identifier.c_str(), aln.size());
                // printf("CPU fallback on No. %d (%s), Alignment Length: %d\n", k, tree->allNodes[nodes[nIdx].first->identifier]->identifier.c_str(), aln.size());
            }
            // if (hostAlnLen[k] <= 0) {
            //     std::vector<std::string> reference, query;
            //     std::vector<int8_t> aln;
            //     for (auto s: tree->allNodes[nodes[nIdx].first->identifier]->msa) reference.push_back(s);
            //     for (auto s: tree->allNodes[nodes[nIdx].second->identifier]->msa) query.push_back(s);
            //     Talco_xdrop::Params talco_params(param.match, param.mismatch, param.gapOpen, param.gapExtend, param.xdrop, param.scoreMode);
            //     Talco_xdrop::Align (
            //         talco_params,
            //         reference,
            //         query,
            //         aln
            //     );
            //     for (int j = 0; j < aln.size(); ++j) {
            //         // std::cout << j << ',' << refIndex << ',' << qryIndex << '\n';
            //         if ((aln[j] & 0xFFFF) == 0) {
            //             for (size_t i=0; i<refNum; i++) alignment[i]        += reference[i][refIndex]; 
            //             for (size_t i=0; i<qryNum; i++) alignment[i+refNum] += query[i][qryIndex];
            //             qryIndex++;refIndex++;
            //         }
            //         else if ((aln[j] & 0xFFFF) == 2) {
            //             for (size_t i=0; i<refNum; i++) alignment[i]        += reference[i][refIndex]; 
            //             for (size_t i=0; i<qryNum; i++) alignment[i+refNum] += '-';
            //                 refIndex++;
            //             }
            //         else {
            //             for (size_t i=0; i<refNum; i++) alignment[i]        += '-'; 
            //             for (size_t i=0; i<qryNum; i++) alignment[i+refNum] += query[i][qryIndex];
            //             qryIndex++;
            //         }
            //     }
            //     tree->allNodes[nodes[nIdx].first->identifier]->msaAln = aln;
            //     printf("CPU fallback on No. %d (%s), Alignment Length: %d\n", k, tree->allNodes[nodes[nIdx].first->identifier]->identifier.c_str(), aln.size());
            // }
            else {
                std::vector<int8_t> aln;
                for (int j = 0; j < hostAlnLen[k]; ++j) {
                    aln.push_back(hostAln[k*2*seqLen+j]);
                    // if ((hostAln[k*2*seqLen+j] & 0xFFFF) == 0) {
                    //     for (size_t i=0; i<refNum; i++) alignment[i] += seqs[refStart+i][refIndex]; 
                    //     for (size_t i=0; i<qryNum; i++) alignment[(i+refNum)] += seqs[qryStart+i][qryIndex];
                    //     qryIndex++;refIndex++;
                    // }
                    // else if ((hostAln[k*2*seqLen+j] & 0xFFFF) == 2) {
                    //     for (size_t i=0; i<refNum; i++) alignment[i] += seqs[refStart+i][refIndex];  
                    //     for (size_t i=0; i<qryNum; i++) alignment[(i+refNum)] += "-"; 
                    //     refIndex++;
                    // }
                    // else {
                    //     for (size_t i=0; i<refNum; i++) alignment[i] += "-"; 
                    //     for (size_t i=0; i<qryNum; i++) alignment[(i+refNum)] += seqs[qryStart+i][qryIndex]; 
                    //     qryIndex++;
                    // }
                    tree->allNodes[nodes[nIdx].first->identifier]->msaAln = aln;
                }
            }
        }     
        
        auto reAlnEnd = std::chrono::high_resolution_clock::now();
        std::chrono::nanoseconds reAlnTime = reAlnEnd - reAlnStart;
        printf("Alignment Time: %d ms\n", reAlnTime.count() / 1000000);
        // free memory
        hipFree(deviceFreq);
        hipFree(deviceAlnLen);
        hipFree(deviceAln);
        hipFree(deviceParam);
        hipFree(deviceSeqInfo);
        hipDeviceSynchronize();
        free(hostFreq);
        free(hostAlnLen);
        free(hostAln);
        free(hostParam);
        free(hostSeqInfo);
        
    }
    for (auto n: nodes) {
        tree->allNodes[n.first->identifier]->msa.clear();
        tree->allNodes[n.first->identifier]->msa.push_back(n.first->identifier);
        tree->allNodes[n.second->identifier]->msa.clear();
        tree->allNodes[n.second->identifier]->msa.push_back(n.second->identifier);    
    }
    return;
}
*/

void msaOnSubtree (Tree* T, msa::utility* util, paritionInfo_t* partition, Params& param) {
    std::vector<std::vector<std::pair<Node*, Node*>>> hier;
    
    for (auto &p: partition->partitionsRoot) {
        std::stack<Node*> msaStack;
        getPostOrderList(p.second.first, msaStack);
        std::vector<std::pair<std::pair<Node*, Node*>, int>> subhier;
        int grpID = p.second.first->grpID;
        getMsaHierachy(subhier, msaStack, grpID, 0);
        for (auto h: subhier) {
            while (hier.size() < h.second+1) {
                std::vector<std::pair<Node*, Node*>> temp;
                hier.push_back(temp);
            }
            hier[h.second].push_back(h.first);
        }
    }
    
    int level = 0;
    for (auto m: hier) {
        auto alnStart = std::chrono::high_resolution_clock::now();
        msaPostOrderTraversal_multigpu(T, m, util, param);
        auto alnEnd = std::chrono::high_resolution_clock::now();
        std::chrono::nanoseconds alnTime = alnEnd - alnStart;
        if (m.size() > 1) std::cout << "Level "<< level << ", aligned " << m.size() << " pairs in " <<  alnTime.count() / 1000000 << " ms\n";
        else              std::cout << "Level "<< level << ", aligned " << m.size() << " pair in " <<  alnTime.count() / 1000000 << " ms\n";
        ++level;
    }
    // Push msa results to roots of sub-subtrees
    for (auto p: partition->partitionsRoot) {
        std::stack<Node*> msaStack;
        getPostOrderList(p.second.first, msaStack);
        std::vector<Node*> msaArray;
        while (!msaStack.empty()) {
            msaArray.push_back(msaStack.top());
            msaStack.pop();
        }
        if (msaArray.back()->msaIdx.size() == 0 && msaArray.size() > 1) {
            if (msaArray.size() == 2) {
                T->allNodes[msaArray.back()->identifier]->msaIdx = msaArray[0]->msaIdx;
                util->seqsLen[msaArray.back()->identifier] = util->seqsLen[msaArray[0]->identifier];
                break;
            }
            for (int m = msaArray.size()-2; m >=0; --m) {
                if (msaArray[m]->msaIdx.size()>0) {
                    T->allNodes[msaArray.back()->identifier]->msaIdx = msaArray[m]->msaIdx;
                    util->seqsLen[msaArray.back()->identifier] = util->seqsLen[msaArray[m]->identifier];
                    break;
                }
            }
        }
    }
    return;
}

void alignSubtrees (Tree* T, Tree* newT, msa::utility* util, Params& param) {
    std::vector<std::pair<Node*, Node*>> type1Aln;
    for (auto n: newT->allNodes) {
        for (auto m: n.second->children) {
            if (newT->allNodes[m->identifier]->grpID == newT->allNodes[n.second->identifier]->grpID) {
                type1Aln.push_back(std::make_pair(T->allNodes[m->identifier], T->allNodes[n.second->identifier]));
            }
        }
    }
    // for (auto n: type1Aln) std::cout << n.first->identifier << ':' << util->seqsLen[n.first->identifier] << 
    //                     ',' << n.second->identifier << ':' << util->seqsLen[n.second->identifier] <<'\n';
    createOverlapMSA(T, type1Aln, util, param);
    return;
}

void mergeSubtrees (Tree* T, Tree* newT, msa::utility* util) {
    std::vector<std::pair<Node*, Node*>> mergePairs;
    for (auto n: newT->allNodes) {
        if (n.second->children.size() > 1) {
            mergePairs.push_back(std::make_pair(n.second, n.second->children[0]));
            for (int i = 1; i < n.second->children.size(); ++i) {
                mergePairs.push_back(std::make_pair(n.second->children[0], n.second->children[i]));
            }
        }
        else if (n.second->children.size() == 1) {
            mergePairs.push_back(std::make_pair(n.second, n.second->children[0]));
        }
    }
    
    std::vector<std::pair<Node*, Node*>> singleLevel;
    std::map<std::string, char> addedNodes;
    
    while (true) {
        auto roundStart = std::chrono::high_resolution_clock::now();
        addedNodes.clear();
        singleLevel.clear();
        for (auto it = mergePairs.begin(); it != mergePairs.end();) {
            Node* a = it->first;
            Node* b = it->second;
            if ((a->parent != nullptr && b->parent != nullptr) &&  
                (addedNodes.find(a->identifier) == addedNodes.end() && addedNodes.find(b->identifier) == addedNodes.end())) {
                singleLevel.push_back(std::make_pair(a, b));
                for (auto id: T->allNodes[a->identifier]->msa) addedNodes[id] = 0;
                for (auto id: T->allNodes[b->identifier]->msa) addedNodes[id] = 0;
                mergePairs.erase(it);
            }
            else {
                ++it;
            }
        }
        bool breakLoop = false;
        if (singleLevel.empty()) {
            for (auto mp: mergePairs) {
                singleLevel.push_back(mp);
            }
            breakLoop = true;
        }
        transitivityMerge(T, newT, singleLevel, util);
        auto roundEnd = std::chrono::high_resolution_clock::now();
        std::chrono::nanoseconds roundTime = roundEnd - roundStart;
        if (singleLevel.size() > 1) {
            std::cout << "Merged "<< singleLevel.size() << " edges in " << roundTime.count() / 1000000 << " ms\n";
        }
        else {
            std::cout << "Merged "<< singleLevel.size() << " edge in " << roundTime.count() / 1000000 << " ms\n";
        }
        if (breakLoop) break;
    }
    return;
}

void createOverlapMSA(Tree* tree, std::vector<std::pair<Node*, Node*>>& nodes, msa::utility* util, Params& param)
{

    int numBlocks = 1024; 
    int blockSize = THREAD_NUM;
    int gpuNum = util->gpuNum;
    // hipGetDeviceCount(&gpuNum); // number of CUDA devices
    
    // get maximum sequence/profile length 
    int32_t seqLen = (util->nowProcess == 0) ? util->memLen : util->seqLen;
    int roundGPU = nodes.size() / numBlocks + 1;
    if (nodes.size()%numBlocks == 0) roundGPU -= 1;
    if (roundGPU < gpuNum) gpuNum = roundGPU;

    paramType* hostParam = (paramType*)malloc(28 * sizeof(paramType)); 

    if (!param.userDefine) {
        for (int i = 0; i < 5; ++i) {
            for (int j = 0; j < 5; ++j) {
                if (i == 5 || j == 5)          hostParam[i*5+j] = 0;
                else if (i == j)               hostParam[i*5+j] = param.match;
                else if (i-j == 2 || j-i == 2) hostParam[i*5+j] = param.trans;
                else                           hostParam[i*5+j] = param.mismatch;
            }
        }
        hostParam[25] = param.gapOpen;
        hostParam[26] = param.gapExtend;
        hostParam[27] = param.xdrop;
    }
    else {
        for (int i = 0; i < 5; ++i) for (int j = 0; j < 5; ++j) hostParam[i*5+j] = param.userMatrix[i][j];
        hostParam[25] = param.userGapOpen;
        hostParam[26] = param.userGapExtend;
        hostParam[27] = param.xdrop;
    }

    std::vector<std::vector<std::pair<int32_t, int32_t>>> seqIdx;
    
    uint16_t** hostFreq = new uint16_t* [gpuNum];
    int8_t**   hostAln = new int8_t* [gpuNum];
    int32_t**  hostLen = new int32_t* [gpuNum];
    int32_t**  hostAlnLen = new int32_t* [gpuNum];
    int32_t**  hostSeqInfo = new int32_t* [gpuNum];

    uint16_t** deviceFreq = new uint16_t* [gpuNum];
    int8_t**   deviceAln = new int8_t* [gpuNum];
    int32_t**  deviceLen = new int32_t* [gpuNum];
    int32_t**  deviceAlnLen = new int32_t* [gpuNum];
    int32_t**  deviceSeqInfo = new int32_t* [gpuNum];
    paramType**  deviceParam = new paramType* [gpuNum];
  
    std::atomic<int> nowRound;
    nowRound.store(0);

    tbb::parallel_for(tbb::blocked_range<int>(0, gpuNum), [&](tbb::blocked_range<int> range){ 
        for (int gn = range.begin(); gn < range.end(); ++gn) {
            hostFreq[gn] = (uint16_t*)malloc(12 * seqLen * numBlocks * sizeof(uint16_t));
            hostAln[gn] = (int8_t*)malloc(    2 * seqLen * numBlocks * sizeof(int8_t));
            hostLen[gn] = (int32_t*)malloc(   2 *          numBlocks * sizeof(int32_t));
            hostAlnLen[gn] = (int32_t*)malloc(             numBlocks * sizeof(int32_t));
            hostSeqInfo[gn] = (int32_t*)malloc(5 * sizeof(int32_t));
            
            hipSetDevice(gn);
            // hipError_t error;
            hipMalloc((void**)&deviceFreq[gn],  12 * seqLen * numBlocks * sizeof(uint16_t));
            hipMalloc((void**)&deviceAln[gn],    2 * seqLen * numBlocks * sizeof(int8_t));
            hipMalloc((void**)&deviceLen[gn],    2 *          numBlocks * sizeof(int32_t));
            hipMalloc((void**)&deviceAlnLen[gn],              numBlocks * sizeof(int32_t));
            hipMalloc((void**)&deviceSeqInfo[gn], 5 * sizeof(int32_t));
            hipMalloc((void**)&deviceParam[gn],  28 * sizeof(paramType));

            hipMemcpy(deviceParam[gn], hostParam, 28 * sizeof(paramType), hipMemcpyHostToDevice);
            // error = hipGetLastError(); printf("CUDA error Malloc: %s\n",hipGetErrorString(error)); 
            std::vector<std::pair<int, int>> seqIdx;
            
            while (nowRound < roundGPU) {
                int rn = nowRound.fetch_add(1);
                int alnPairs = (nodes.size() - rn*numBlocks > numBlocks) ? numBlocks : nodes.size() - rn*numBlocks;
                // int seqNum = 0;
                
                // Initailize 
                for (int n = 0; n < 12*seqLen * numBlocks; ++n) hostFreq[gn][n] = 0;
                for (int n = 0; n <  2*seqLen * numBlocks; ++n) hostAln[gn][n] = 0;
                for (int n = 0; n <  2*         numBlocks; ++n) hostLen[gn][n] = 0;
                for (int n = 0; n <             numBlocks; ++n) hostAlnLen[gn][n] = 0;
                seqIdx.clear();

                // Calculate Frequency
                for (int n = 0; n < alnPairs; ++n) {
                    int32_t nIdx = n + rn*numBlocks;
                    // int32_t qryIdx = 0;
                    // int32_t refIdx = 0;
                    int32_t refLen = util->seqsLen[nodes[nIdx].first->identifier];
                    int32_t qryLen = util->seqsLen[nodes[nIdx].second->identifier];
                    if (util->nowProcess == 0) {
                        // refIdx = seqNum;
                        for (auto sIdx: tree->allNodes[nodes[nIdx].first->identifier]->msaIdx) { 
                            int storage = util->seqsStorage[sIdx];
                            tbb::parallel_for(tbb::blocked_range<int>(0, refLen), [&](tbb::blocked_range<int> r) {
                            for (int s = r.begin(); s < r.end(); ++s) {
                            // for (int s = 0; s < refLen; ++s) {
                                if      (util->alnStorage[storage][sIdx][s] == 'A' || util->alnStorage[storage][sIdx][s] == 'a') hostFreq[gn][12*seqLen*n+6*s+0]+=1;
                                else if (util->alnStorage[storage][sIdx][s] == 'C' || util->alnStorage[storage][sIdx][s] == 'c') hostFreq[gn][12*seqLen*n+6*s+1]+=1;
                                else if (util->alnStorage[storage][sIdx][s] == 'G' || util->alnStorage[storage][sIdx][s] == 'g') hostFreq[gn][12*seqLen*n+6*s+2]+=1;
                                else if (util->alnStorage[storage][sIdx][s] == 'T' || util->alnStorage[storage][sIdx][s] == 't' ||
                                         util->alnStorage[storage][sIdx][s] == 'U' || util->alnStorage[storage][sIdx][s] == 'u') hostFreq[gn][12*seqLen*n+6*s+3]+=1;
                                else if (util->alnStorage[storage][sIdx][s] == 'N' || util->alnStorage[storage][sIdx][s] == 'n') hostFreq[gn][12*seqLen*n+6*s+4]+=1;
                                else                                                                                             hostFreq[gn][12*seqLen*n+6*s+5]+=1;
                            }
                            });
                            // seqNum += 1;
                        }
                        // qryIdx = seqNum;
                        for (auto sIdx: tree->allNodes[nodes[nIdx].second->identifier]->msaIdx) { 
                            int storage = util->seqsStorage[sIdx];
                            tbb::parallel_for(tbb::blocked_range<int>(0, qryLen), [&](tbb::blocked_range<int> r) {
                            for (int s = r.begin(); s < r.end(); ++s) {
                            // for (int s = 0; s < qryLen; ++s) {
                                if      (util->alnStorage[storage][sIdx][s] == 'A' || util->alnStorage[storage][sIdx][s] == 'a') hostFreq[gn][12*seqLen*n+6*(seqLen+s)+0]+=1;
                                else if (util->alnStorage[storage][sIdx][s] == 'C' || util->alnStorage[storage][sIdx][s] == 'c') hostFreq[gn][12*seqLen*n+6*(seqLen+s)+1]+=1;
                                else if (util->alnStorage[storage][sIdx][s] == 'G' || util->alnStorage[storage][sIdx][s] == 'g') hostFreq[gn][12*seqLen*n+6*(seqLen+s)+2]+=1;
                                else if (util->alnStorage[storage][sIdx][s] == 'T' || util->alnStorage[storage][sIdx][s] == 't' ||
                                         util->alnStorage[storage][sIdx][s] == 'U' || util->alnStorage[storage][sIdx][s] == 'u') hostFreq[gn][12*seqLen*n+6*(seqLen+s)+3]+=1;
                                else if (util->alnStorage[storage][sIdx][s] == 'N' || util->alnStorage[storage][sIdx][s] == 'n') hostFreq[gn][12*seqLen*n+6*(seqLen+s)+4]+=1;
                                else                                                                                             hostFreq[gn][12*seqLen*n+6*(seqLen+s)+5]+=1;
                            }
                            });
                            // seqNum += 1;
                        }
                    }
                    else {
                        int subtreeRef = tree->allNodes[nodes[nIdx].first->identifier]->grpID;
                        for (int i = 0; i < 6; ++i) {
                            for (int s = 0; s < refLen; ++s) {
                                hostFreq[gn][12*seqLen*n+6*s+i] = util->profileFreq[subtreeRef][i][s]; 
                            }
                        }
                        int subtreeQry = tree->allNodes[nodes[nIdx].second->identifier]->grpID;
                        for (int i = 0; i < 6; ++i) {
                            for (int s = 0; s < qryLen; ++s) {
                                hostFreq[gn][12*seqLen*n+6*(seqLen+s)+i] = util->profileFreq[subtreeQry][i][s]; 
                            }
                        }
                    }
                    hostLen[gn][2*n] = refLen; hostLen[gn][2*n+1] = qryLen;
                    // seqIdx.push_back(std::make_pair(refIdx, qryIdx));
                }


                hostSeqInfo[gn][0] = seqLen;
                hostSeqInfo[gn][1] = 0;
                hostSeqInfo[gn][2] = alnPairs;
                hostSeqInfo[gn][3] = numBlocks;
                hostSeqInfo[gn][4] = param.userDefine;
        
                hipMemcpy(deviceFreq[gn], hostFreq[gn], 12*seqLen * numBlocks * sizeof(uint16_t), hipMemcpyHostToDevice);
                hipMemcpy(deviceAln[gn], hostAln[gn], 2*seqLen * numBlocks * sizeof(int8_t), hipMemcpyHostToDevice);
                hipMemcpy(deviceLen[gn], hostLen[gn], 2*numBlocks * sizeof(int32_t), hipMemcpyHostToDevice);
                hipMemcpy(deviceAlnLen[gn], hostAlnLen[gn], numBlocks * sizeof(int32_t), hipMemcpyHostToDevice);
                hipMemcpy(deviceSeqInfo[gn], hostSeqInfo[gn], 5 * sizeof(int32_t), hipMemcpyHostToDevice);
                
                std::string berr = hipGetErrorString(hipGetLastError());
                if (berr != "no error") printf("ERROR: Before kernel %s!\n", berr.c_str());
                alignGrpToGrp_talco<<<numBlocks, blockSize>>>(
                    deviceFreq[gn],
                    deviceAln[gn], 
                    deviceLen[gn],
                    deviceAlnLen[gn],
                    deviceSeqInfo[gn], 
                    deviceParam[gn]
                );
                hipDeviceSynchronize();
                std::string aerr = hipGetErrorString(hipGetLastError());
                if (aerr != "no error") printf("ERROR: After kernel %s!\n", aerr.c_str());
                
                hipMemcpy(hostAln[gn], deviceAln[gn], 2*seqLen * numBlocks * sizeof(int8_t), hipMemcpyDeviceToHost);
                hipMemcpy(hostAlnLen[gn], deviceAlnLen[gn], numBlocks * sizeof(int32_t), hipMemcpyDeviceToHost);
                hipDeviceSynchronize();
                // int maxAlnLen = 0;
                // for (int n = 0; n <  alnPairs; ++n) {
                //     if (hostAlnLen[gn][n] > maxAlnLen) maxAlnLen = hostAlnLen[gn][n];
                // }
                // util->memCheck(maxAlnLen);
                
                
                // tbb::parallel_for(tbb::blocked_range<int>(0, alignSize[gn]), [&](tbb::blocked_range<int> range) {
                // for (int k = range.begin(); k < range.end(); ++k) {
                for (int n = 0; n < alnPairs; ++n) {
                    // int32_t refNum = seqIdx[n].second - seqIdx[n].first;
                    // int32_t qryNum = (n !=  alnPairs-1) ? seqIdx[n+1].first - seqIdx[n].second : seqNum - seqIdx[n].second;
                    int32_t nIdx = n + rn*numBlocks;
                    if (hostAlnLen[gn][n] <= 0) {
                        int32_t refLen = util->seqsLen[nodes[nIdx].first->identifier];
                        int32_t qryLen = util->seqsLen[nodes[nIdx].second->identifier];
                        uint16_t *freq = new uint16_t[12*seqLen]; 
                        for (int i = 0; i < 12*seqLen; ++i) freq[i] = hostFreq[gn][12*seqLen*n+i];
                        std::vector<int8_t> aln;
                        alignGrpToGrp_traditional (
                            freq,
                            seqLen,
                            refLen,
                            qryLen,
                            param,
                            aln
                        );
                        delete [] freq;
                        int32_t alnLen = aln.size();
                        util->memCheck(alnLen);
                        std::reverse(aln.begin(), aln.end());
                        tree->allNodes[nodes[nIdx].first->identifier]->msaAln = aln;
                        std::cout << "CPU fallback (traditional global alignment) on No. " << n << " (" << tree->allNodes[nodes[nIdx].first->identifier]->identifier << ")\n";
                    }
                    else {
                        std::vector<int8_t> aln;
                        for (int j = 0; j < hostAlnLen[gn][n]; ++j) {
                            aln.push_back(hostAln[gn][n*2*seqLen+j]);
                        }
                        tree->allNodes[nodes[nIdx].first->identifier]->msaAln = aln;
                    }
                }    
            }  
        }
    });
    
    for (auto n: nodes) {
        tree->allNodes[n.first->identifier]->msa.clear();
        tree->allNodes[n.first->identifier]->msa.push_back(n.first->identifier);
        tree->allNodes[n.second->identifier]->msa.clear();
        tree->allNodes[n.second->identifier]->msa.push_back(n.second->identifier);    
    }

    // free memory  
    for (int gn = 0; gn < gpuNum; ++gn) {
        hipSetDevice(gn);
        hipFree(deviceFreq[gn]);
        hipFree(deviceAlnLen[gn]);
        hipFree(deviceLen[gn]);
        hipFree(deviceAln[gn]);
        hipFree(deviceSeqInfo[gn]);
        hipFree(deviceParam[gn]);
        hipDeviceSynchronize();  
        free(hostFreq[gn]);
        free(hostAlnLen[gn]);
        free(hostLen[gn]);
        free(hostAln[gn]);
        free(hostSeqInfo[gn]);
    }
    
    free(hostParam);

    delete [] deviceFreq;
    delete [] deviceAlnLen;
    delete [] deviceAln;
    delete [] deviceParam;
    delete [] deviceSeqInfo;
    delete [] hostFreq;
    delete [] hostAlnLen;
    delete [] hostLen;
    delete [] hostAln;
    delete [] hostSeqInfo;
    return;
}

void transitivityMerge(Tree* tree, Tree* newtree, std::vector<std::pair<Node*, Node*>>& nodes, msa::utility* util) {
    
    tbb::parallel_for(tbb::blocked_range<int>(0, nodes.size()), [&](tbb::blocked_range<int> range) {
    for (int s = range.begin(); s < range.end(); ++s) {
        auto n = nodes[s];
    // for (auto n: nodes) {
        if (newtree->allNodes[n.first->identifier]->parent == nullptr) {
            
            for (auto id: tree->allNodes[n.second->identifier]->msa) tree->allNodes[n.first->identifier]->msa.push_back(id);
            std::vector<int8_t> rootAln;
            for (int i = 0; i < tree->allNodes[n.second->identifier]->msaAln.size(); ++i) {
                if ((tree->allNodes[n.second->identifier]->msaAln[i] & 0XFFFF) == 2) rootAln.push_back(1);
                else if ((tree->allNodes[n.second->identifier]->msaAln[i] & 0XFFFF) == 1) rootAln.push_back(2);
                else rootAln.push_back(tree->allNodes[n.second->identifier]->msaAln[i]);
            }
            tree->allNodes[n.first->identifier]->msaAln = rootAln;

            int seqLen = tree->allNodes[n.first->identifier]->msaAln.size();
            util->seqsLen[n.first->identifier] = seqLen;
            if (util->nowProcess == 0) {
                util->memCheck(seqLen);
                for (auto id: tree->allNodes[n.first->identifier]->msa) {
                    // auto id = tree->root->msa[k];
                    std::vector<int8_t> aln = tree->allNodes[id]->msaAln;
                    for (auto sIdx: tree->allNodes[id]->msaIdx) {
                        int orgIdx = 0;
                        int storeFrom = util->seqsStorage[sIdx];
                        int storeTo = 1 - util->seqsStorage[sIdx];
                        for (int j = 0; j < aln.size(); ++j) {
                            if ((aln[j] & 0xFFFF) == 0 || (aln[j] & 0xFFFF) == 2) {
                                util->alnStorage[storeTo][sIdx][j] = util->alnStorage[storeFrom][sIdx][orgIdx];
                                orgIdx++;
                            }
                            else {
                                util->alnStorage[storeTo][sIdx][j] = '-';
                            }
                        }
                        util->seqsLen[id] = aln.size();
                        util->changeStorage(sIdx);
                    }
                }
                // });
                for (auto id: tree->allNodes[n.first->identifier]->msa) {
                    if (id != tree->allNodes[n.first->identifier]->identifier) {
                        for (auto Idx: tree->allNodes[id]->msaIdx) {
                            tree->allNodes[n.first->identifier]->msaIdx.push_back(Idx);
                        }
                    }
                }
                tree->allNodes[n.first->identifier]->msa.clear();
                continue;
            }
            else {
                util->seqLen = seqLen;
                if (util->seqNum != 0) {
                    for (auto id: tree->allNodes[n.first->identifier]->msa) {
                        std::vector<int8_t> aln = tree->allNodes[id]->msaAln;
                        for (auto sIdx: tree->allNodes[id]->msaIdx) {
                            char* seq = new char[seqLen+1];
                            int orgIdx = 0;
                            for (int j = 0; j < seqLen+1; ++j) {
                                if (j < aln.size()) {
                                    if ((aln[j] & 0xFFFF) == 0 || (aln[j] & 0xFFFF) == 2) {
                                        seq[j] = util->seqs[sIdx][orgIdx];
                                        orgIdx++;
                                    }
                                    else {
                                        seq[j] = '-';
                                    }
                                }
                                else {
                                    seq[j] = 0;
                                }

                            }
                            util->seqsLen[id] = aln.size();
                            // util->changeStorage(sIdx);
                            delete [] util->seqs[sIdx];
                            util->seqs[sIdx] = seq;
                        }
                    }
                    // });
                    for (auto id: tree->allNodes[n.first->identifier]->msa) {
                        if (id != tree->allNodes[n.first->identifier]->identifier) {
                            for (auto Idx: tree->allNodes[id]->msaIdx) {
                                tree->allNodes[n.first->identifier]->msaIdx.push_back(Idx);
                            }
                        }
                    }
                    tree->allNodes[n.first->identifier]->msa.clear();
                
                }
                continue;
            }
            
        }
        int8_t refGap = (newtree->allNodes[n.first->identifier]->level == newtree->allNodes[n.second->identifier]->level) ? 2 : 1; 
        std::vector<std::vector<int8_t>> refAln, qryAln;
        std::vector<std::vector<int8_t>> refNewAln, qryNewAln;
        // refAln.push_back(tree->allNodes[n.first->identifier]->msaAln);
        // qryAln.push_back(tree->allNodes[n.second->identifier]->msaAln);
        for (auto id: tree->allNodes[n.first->identifier]->msa)  refAln.push_back(tree->allNodes[id]->msaAln);
        for (auto id: tree->allNodes[n.second->identifier]->msa) qryAln.push_back(tree->allNodes[id]->msaAln);
        
        std::vector<int8_t> refOpAln = tree->allNodes[n.first->identifier]->msaAln;
        std::vector<int8_t> qryOpAln = tree->allNodes[n.second->identifier]->msaAln;
        int32_t refLen = refOpAln.size();
        int32_t qryLen = qryOpAln.size();
        int32_t refNum = refAln.size();
        int32_t qryNum = qryAln.size();
        int32_t seqLen = max(refLen, qryLen);
        for (int i = 0; i < refNum; ++i) {
            std::vector<int8_t> temp;
            refNewAln.push_back(temp);
        }
        for (int i = 0; i < qryNum; ++i) {
            std::vector<int8_t> temp;
            qryNewAln.push_back(temp);
        }
        // std::cout << refLen << ':';
        // for (int i = 0; i < refLen; ++i) {
        //     if ((refOpAln[i] & 0XFFFF) == refGap || (refOpAln[i] & 0XFFFF) == 3) 
        //         std::cout << i << ',';
        // }
        // std::cout << '\n';
        // std::cout << qryLen << ':';
        // for (int i = 0; i < qryLen; ++i) {
        //     if ((qryOpAln[i] & 0XFFFF) == 2 || (qryOpAln[i] & 0XFFFF) == 3) 
        //         std::cout << i << ',';
        // }
        // std::cout << '\n';
        // aln = 0: match
        // aln = 1: ref gap
        // aln = 2: qry gap
        // aln = 3: permenant gap
        int32_t rIdx = 0, qIdx = 0;
        while (rIdx < refLen && qIdx < qryLen) {
            if (((refOpAln[rIdx] & 0xFFFF) != refGap && (refOpAln[rIdx] & 0xFFFF) != 3)  &&
                ((qryOpAln[qIdx] & 0xFFFF) != 2 && (qryOpAln[qIdx] & 0xFFFF) != 3)) {
                for (size_t i=0; i<refNum; i++)      refNewAln[i].push_back(refAln[i][rIdx]);
                for (size_t i=0; i<qryNum; i++)      qryNewAln[i].push_back(qryAln[i][qIdx]);
                qIdx++;rIdx++;
            }
            else if (((refOpAln[rIdx] & 0xFFFF) == refGap || (refOpAln[rIdx] & 0xFFFF) == 3)  &&
                     ((qryOpAln[qIdx] & 0xFFFF) != 2 && (qryOpAln[qIdx] & 0xFFFF) != 3)) {
                int consecGap = 0;
                int k = rIdx;
                while (((refOpAln[k] & 0xFFFF) == refGap || (refOpAln[k] & 0xFFFF) == 3) && k < refLen) {
                    ++consecGap;
                    ++k;
                }
                for (size_t g = 0; g < consecGap; ++g) {
                    for (size_t i=0; i<refNum; i++)      refNewAln[i].push_back(refAln[i][rIdx]);
                    for (size_t i=0; i<qryNum; i++)      qryNewAln[i].push_back(3);
                    rIdx += 1;
                }
            }
            else if (((refOpAln[rIdx] & 0xFFFF) != refGap && (refOpAln[rIdx] & 0xFFFF) != 3)  &&
                     ((qryOpAln[qIdx] & 0xFFFF) == 2 || (qryOpAln[qIdx] & 0xFFFF) == 3)) {
                int consecGap = 0;
                int k = qIdx;
                while (((qryOpAln[k] & 0xFFFF) == 2 || (qryOpAln[k] & 0xFFFF) == 3) && k < qryLen) {
                    ++consecGap;
                    ++k;
                }
                for (size_t g = 0; g < consecGap; ++g) {
                    for (size_t i=0; i<refNum; i++)      refNewAln[i].push_back(3);
                    for (size_t i=0; i<qryNum; i++)      qryNewAln[i].push_back(qryAln[i][qIdx]);
                    qIdx += 1;
                }
            }
            else {
                int consecGap = 0;
                int kr = rIdx, kq = qIdx;
                while (((refOpAln[kr] & 0xFFFF) == refGap || (refOpAln[kr] & 0xFFFF) == 3) && kr < refLen) {
                    ++consecGap;
                    ++kr;
                }
                for (size_t g = 0; g < consecGap; ++g) {
                    for (size_t i=0; i<refNum; i++)      refNewAln[i].push_back(refAln[i][rIdx]);
                    for (size_t i=0; i<qryNum; i++)      qryNewAln[i].push_back(3);
                    rIdx += 1;
                }
                consecGap = 0;
                while (((qryOpAln[kq] & 0xFFFF) == 2 || (qryOpAln[kq] & 0xFFFF) == 3) && kq < qryLen) {
                    ++consecGap;
                    ++kq;
                }
                for (size_t g = 0; g < consecGap; ++g) {
                    for (size_t i=0; i<refNum; i++)      refNewAln[i].push_back(3);
                    for (size_t i=0; i<qryNum; i++)      qryNewAln[i].push_back(qryAln[i][qIdx]);
                    qIdx += 1;
                }
            }
        }
        // printf("rIdx:%d, qIdx:%d, refLen:%d, qryLen:%d, alnLen: %d\n", rIdx, qIdx, refLen, qryLen, alignment[0].size());
        if (rIdx < refLen) {
            for (size_t g = rIdx; g < refLen; ++g) {
                for (size_t i=0; i<refNum; i++)      refNewAln[i].push_back(refAln[i][g]);
                for (size_t i=0; i<qryNum; i++)      qryNewAln[i].push_back(3);    
            }
        }
        if (qIdx < qryLen) {
            for (size_t g = qIdx; g < qryLen; ++g) {
                for (size_t i=0; i<refNum; i++)      refNewAln[i].push_back(3);
                for (size_t i=0; i<qryNum; i++)      qryNewAln[i].push_back(qryAln[i][g]);
            }
        }
        assert (refNewAln[0].size() == qryNewAln[0].size());
        for (int i = 0; i < tree->allNodes[n.first->identifier]->msa.size(); ++i) {
            std::string id = tree->allNodes[n.first->identifier]->msa[i];
            tree->allNodes[id]->msaAln = refNewAln[i];
        } 
        for (int i = 0; i < tree->allNodes[n.second->identifier]->msa.size(); ++i) {
            std::string id = tree->allNodes[n.second->identifier]->msa[i];
            tree->allNodes[id]->msaAln = qryNewAln[i];
        } 
        
        std::vector<std::string> temp = tree->allNodes[n.first->identifier]->msa;

        for (int r = 1; r < tree->allNodes[n.first->identifier]->msa.size(); ++r) {
            std::string grpNode = tree->allNodes[n.first->identifier]->msa[r];
            for (auto id: tree->allNodes[n.second->identifier]->msa) tree->allNodes[grpNode]->msa.push_back(id);
        }
        for (auto id: tree->allNodes[n.second->identifier]->msa) tree->allNodes[n.first->identifier]->msa.push_back(id);
        for (int r = 1; r < tree->allNodes[n.second->identifier]->msa.size(); ++r) {
            std::string grpNode = tree->allNodes[n.second->identifier]->msa[r];
            for (auto id: temp) tree->allNodes[grpNode]->msa.push_back(id);
        }
        for (auto id: temp) tree->allNodes[n.second->identifier]->msa.push_back(id);
    }
    });
    
    return;
}

/*
void msaPostOrderTraversal_multigpu(Tree* tree, std::vector<std::pair<Node*, Node*>> nodes, msa::utility* util, Params& param)
{

    auto freqStart = std::chrono::high_resolution_clock::now();
    for (auto n_pair: nodes) {
        auto n = std::make_pair(tree->allNodes[n_pair.first->identifier], tree->allNodes[n_pair.second->identifier]);
        if (n.first->children.size()==0) {
            tree->allNodes[n.first->identifier]->msaIdx.push_back(util->seqsIdx[n.first->identifier]);
        }
        else {
            if (tree->allNodes[n.first->identifier]->msaIdx.size() == 0) {
                Node* node = tree->allNodes[n.first->identifier];
                int grpID = node->grpID;
                for (int childIndex=0; childIndex<node->children.size(); childIndex++) {
                    if ((node->children[childIndex]->grpID == -1 || node->children[childIndex]->grpID == grpID) && (node->children[childIndex]->identifier != n.second->identifier)) {
                        if (node->children[childIndex]->msaIdx.size() == 0) tree->allNodes[node->children[childIndex]->identifier]->msaIdx.push_back(util->seqsIdx[node->children[childIndex]->identifier]);
                        tree->allNodes[n.first->identifier]->msaIdx = node->children[childIndex]->msaIdx;
                        util->seqsLen[n.first->identifier] = util->seqsLen[node->children[childIndex]->identifier];
                        break;
                    }
                }
            }
        }
        if (n.second->children.size()==0) {
            tree->allNodes[n.second->identifier]->msaIdx.push_back(util->seqsIdx[n.second->identifier]);
        }
        else {
            if (tree->allNodes[n.second->identifier]->msaIdx.size() == 0) {
                Node* node = tree->allNodes[n.second->identifier];
                int grpID = node->grpID;
                for (int childIndex=0; childIndex<node->children.size(); childIndex++) {
                    if ((node->children[childIndex]->grpID == -1 || node->children[childIndex]->grpID == grpID) && (node->children[childIndex]->identifier != n.first->identifier)) {
                        if (node->children[childIndex]->msaIdx.size() == 0) tree->allNodes[node->children[childIndex]->identifier]->msaIdx.push_back(util->seqsIdx[node->children[childIndex]->identifier]);
                        tree->allNodes[n.second->identifier]->msaIdx = node->children[childIndex]->msaIdx;
                        util->seqsLen[n.second->identifier] = util->seqsLen[node->children[childIndex]->identifier];
                        break;
                    }
                }
            }
        }
        // for (auto k: tree->allNodes[n.first->identifier]->msaIdx) std::cout << k << ',';
        // std::cout << '\n';
        // for (auto k: tree->allNodes[n.second->identifier]->msaIdx) std::cout << k << ',';
        // std::cout << '\n';
    }
    // });

    int numBlocks = 1024; 
    int blockSize = THREAD_NUM;
    int gpuNum;
    hipGetDeviceCount(&gpuNum); // number of CUDA devices
    
    // get maximum sequence/profile length 
    int32_t seqLen = 0;
    for (auto n: nodes) {
        int32_t refLen = util->seqsLen[n.first->identifier];
        int32_t qryLen = util->seqsLen[n.second->identifier];
        int32_t tempMax = max(qryLen, refLen);
        seqLen = max(seqLen, tempMax);
    }
    
    int roundGPU = nodes.size() / numBlocks + 1;
    if (nodes.size()%numBlocks == 0) roundGPU -= 1;
    if (roundGPU < gpuNum) gpuNum = roundGPU;

    int* alignSize = new int[roundGPU];
    int32_t* seqNum = new int32_t[roundGPU];
    uint16_t** hostFreq = new uint16_t* [roundGPU];
    int8_t**   hostAln = new int8_t* [roundGPU];
    int32_t**  hostLen = new int32_t* [roundGPU];
    int32_t**  hostAlnLen = new int32_t* [roundGPU];
    int32_t**  hostSeqInfo = new int32_t* [roundGPU];
    
    paramType* hostParam = (paramType*)malloc(28 * sizeof(paramType)); 

    if (param.scoreMode == 0) {
        for (int i = 0; i < 5; ++i) {
            for (int j = 0; j < 5; ++j) {
                if (i == 5 || j == 5)          hostParam[i*5+j] = 0;
                else if (i == j)               hostParam[i*5+j] = param.match;
                else if (i-j == 2 || j-i == 2) hostParam[i*5+j] = param.trans;
                else                           hostParam[i*5+j] = param.mismatch;
            }
        }
        hostParam[25] = param.gapOpen;
        hostParam[26] = param.gapExtend;
        hostParam[27] = param.xdrop;
    }
    else if (param.scoreMode == 1) {
        for (int i = 0; i < 5; ++i) for (int j = 0; j < 5; ++j) hostParam[i*5+j] = param.hoxd70[i][j];
        hostParam[25] = param.hoxd70_gapOpen;
        hostParam[26] = param.hoxd70_gapExtend;
        hostParam[27] = param.xdrop;
    }
    
    
    std::vector<std::vector<uint16_t*>> freq;
    std::vector<std::vector<std::pair<int32_t, int32_t>>> seqIdx;
    std::vector<std::vector<std::pair<int32_t, int32_t>>> len;
    for (int rn = 0; rn < roundGPU; ++rn) {
        int pairsLeft = nodes.size() - rn*numBlocks;
        if (pairsLeft < numBlocks) alignSize[rn] = pairsLeft;
        else alignSize[rn] = numBlocks;
        seqNum[rn] = 0;
        hostFreq[rn] = (uint16_t*)malloc(12*seqLen * alignSize[rn] * sizeof(uint16_t));
        hostAln[rn] = (int8_t*)malloc(2*seqLen * alignSize[rn] * sizeof(int8_t));
        hostLen[rn] = (int32_t*)malloc(2*alignSize[rn] * sizeof(int32_t));
        hostAlnLen[rn] = (int32_t*)malloc(alignSize[rn] * sizeof(int32_t));
        hostSeqInfo[rn] = (int32_t*)malloc(5 * sizeof(int32_t));
        // store all sequences to array
        std::vector<uint16_t*> freqTemp;
        std::vector<std::pair<int32_t, int32_t>> seqIdxTemp;
        std::vector<std::pair<int32_t, int32_t>> lenTemp;
        for (int n = 0; n < alignSize[rn]; ++n) {
            int32_t nIdx = n + rn*numBlocks;
            int32_t qryIdx = 0;
            int32_t refIdx = 0;
            int32_t refLen = util->seqsLen[nodes[nIdx].first->identifier];
            int32_t qryLen = util->seqsLen[nodes[nIdx].second->identifier];
            refIdx = seqNum[rn];
            uint16_t *temp = new uint16_t[12*seqLen]; 
            for (int i = 0; i < 12*seqLen; ++i) temp[i]=0;
            // assert(temp.size() == 12*seqLen);
            // tbb::blocked_range<int> rangeRef(0, refLen);
            for (auto sIdx: tree->allNodes[nodes[nIdx].first->identifier]->msaIdx) { 
                int storage = util->seqsStorage[sIdx];
                tbb::parallel_for(tbb::blocked_range<int>(0, refLen), [&](tbb::blocked_range<int> r) {
                for (int s = r.begin(); s < r.end(); ++s) {
                    if      (util->seqBuf[storage][sIdx][s] == 'A' || util->seqBuf[storage][sIdx][s] == 'a') temp[6*s+0]+=1;
                    else if (util->seqBuf[storage][sIdx][s] == 'C' || util->seqBuf[storage][sIdx][s] == 'c') temp[6*s+1]+=1;
                    else if (util->seqBuf[storage][sIdx][s] == 'G' || util->seqBuf[storage][sIdx][s] == 'g') temp[6*s+2]+=1;
                    else if (util->seqBuf[storage][sIdx][s] == 'T' || util->seqBuf[storage][sIdx][s] == 't' ||
                             util->seqBuf[storage][sIdx][s] == 'U' || util->seqBuf[storage][sIdx][s] == 'u') temp[6*s+3]+=1;
                    else if (util->seqBuf[storage][sIdx][s] == 'N' || util->seqBuf[storage][sIdx][s] == 'n') temp[6*s+4]+=1;
                    else                                                                                     temp[6*s+5]+=1;
                }
                });
                seqNum[rn] += 1;
            }
            qryIdx = seqNum[rn];
            for (auto sIdx: tree->allNodes[nodes[nIdx].second->identifier]->msaIdx) { 
                int storage = util->seqsStorage[sIdx];
                tbb::parallel_for(tbb::blocked_range<int>(0, qryLen), [&](tbb::blocked_range<int> r) {
                for (int s = r.begin(); s < r.end(); ++s) {
                    if      (util->seqBuf[storage][sIdx][s] == 'A' || util->seqBuf[storage][sIdx][s] == 'a') temp[6*(seqLen+s)+0]+=1;
                    else if (util->seqBuf[storage][sIdx][s] == 'C' || util->seqBuf[storage][sIdx][s] == 'c') temp[6*(seqLen+s)+1]+=1;
                    else if (util->seqBuf[storage][sIdx][s] == 'G' || util->seqBuf[storage][sIdx][s] == 'g') temp[6*(seqLen+s)+2]+=1;
                    else if (util->seqBuf[storage][sIdx][s] == 'T' || util->seqBuf[storage][sIdx][s] == 't' ||
                             util->seqBuf[storage][sIdx][s] == 'U' || util->seqBuf[storage][sIdx][s] == 'u') temp[6*(seqLen+s)+3]+=1;
                    else if (util->seqBuf[storage][sIdx][s] == 'N' || util->seqBuf[storage][sIdx][s] == 'n') temp[6*(seqLen+s)+4]+=1;
                    else                                                                     temp[6*(seqLen+s)+5]+=1;
                }
                });
                seqNum[rn] += 1;
            }
            seqIdxTemp.push_back(std::make_pair(refIdx, qryIdx));
            lenTemp.push_back(std::make_pair(refLen, qryLen));
            freqTemp.push_back(temp);
        }
        freq.push_back(freqTemp);
        len.push_back(lenTemp);
        seqIdx.push_back(seqIdxTemp);
    }

    tbb::parallel_for(tbb::blocked_range<int>(0, roundGPU), [&](tbb::blocked_range<int> range){
        for (int gn = range.begin(); gn < range.end(); ++gn) {
            for (int j = 0; j < 2*alignSize[gn]; ++j) { 
                if (j%2 == 0) hostLen[gn][j] = len[gn][j/2].first;
                else          hostLen[gn][j] = len[gn][j/2].second;
            }
            for (int j = 0; j < alignSize[gn]; ++j) {
                for (int l = 0; l < 12*seqLen; ++l) {
                    hostFreq[gn][12*seqLen*j+l] = freq[gn][j][l];
                }
            }
            for (int j = 0; j < 2*seqLen*alignSize[gn]; ++j) { 
                hostAln[gn][j] = 0;
            }
            for (int j = 0; j < alignSize[gn]; ++j) { 
                hostAlnLen[gn][j] = 0;
            }
            hostSeqInfo[gn][0] = seqLen;
            hostSeqInfo[gn][1] = seqNum[gn];
            hostSeqInfo[gn][2] = alignSize[gn];
            hostSeqInfo[gn][3] = numBlocks;
            hostSeqInfo[gn][4] = param.scoreMode;
        }
    });

    auto freqEnd = std::chrono::high_resolution_clock::now();
    std::chrono::nanoseconds freqTime = freqEnd -freqStart;
    printf("Preprocessing time : %d ms\n",  (freqTime.count() / 1000000));        

    auto kernelStart = std::chrono::high_resolution_clock::now();
    uint16_t** deviceFreq = new uint16_t* [gpuNum];
    int8_t**   deviceAln = new int8_t* [gpuNum];
    int32_t**  deviceLen = new int32_t* [gpuNum];
    int32_t**  deviceAlnLen = new int32_t* [gpuNum];
    int32_t**  deviceSeqInfo = new int32_t* [gpuNum];
    paramType**  deviceParam = new paramType* [gpuNum];
    // int nowRound = 0;
    std::atomic<int> nowRound;
    nowRound.store(0);
    tbb::parallel_for(tbb::blocked_range<int>(0, gpuNum), [&](tbb::blocked_range<int> range){ 
        for (int gn = range.begin(); gn < range.end(); ++gn) {
            hipSetDevice(gn);
            int nowMemSize = alignSize[gn];
            // hipError_t error;
            hipMalloc((void**)&deviceFreq[gn], 12*seqLen * alignSize[gn] * sizeof(uint16_t));
            // error = hipGetLastError(); printf("CUDA error Freq1: %s, %d\n, E",hipGetErrorString(error), gn); 
            hipMalloc((void**)&deviceAln[gn], 2*seqLen * alignSize[gn] * sizeof(int8_t));
            // error = hipGetLastError(); printf("CUDA error Freq2: %s\n",hipGetErrorString(error)); 
            hipMalloc((void**)&deviceLen[gn], 2*alignSize[gn] * sizeof(int32_t));
            // error = hipGetLastError(); printf("CUDA error Freq3: %s\n",hipGetErrorString(error)); 
            hipMalloc((void**)&deviceAlnLen[gn], alignSize[gn] * sizeof(int32_t));
            // error = hipGetLastError(); printf("CUDA error Freq4: %s\n",hipGetErrorString(error)); 
            hipMalloc((void**)&deviceSeqInfo[gn], 5 * sizeof(int32_t));
            // error = hipGetLastError(); printf("CUDA error Freq5: %s\n",hipGetErrorString(error)); 
            hipMalloc((void**)&deviceParam[gn], 28 * sizeof(paramType));
            hipMemcpy(deviceParam[gn], hostParam, 28 * sizeof(paramType), hipMemcpyHostToDevice);
            // error = hipGetLastError(); printf("CUDA error Freq6: %s\n",hipGetErrorString(error)); 
                
            while (nowRound < roundGPU) {
                int rn = nowRound.fetch_add(1);
                if (alignSize[rn] != nowMemSize) {
                    // hipSetDevice(gn);
                    hipFree(deviceFreq[gn]);
                    hipFree(deviceAln[gn]);
                    hipFree(deviceLen[gn]);
                    hipFree(deviceAlnLen[gn]);
                    // hipFree(deviceParam[gn]);
                    // hipFree(deviceSeqInfo[gn]);
                    // error = hipGetLastError(); printf("CUDA error Free: %s\n",hipGetErrorString(error)); 
                    hipDeviceSynchronize();
                    hipMalloc((void**)&deviceFreq[gn], 12*seqLen*alignSize[rn] * sizeof(uint16_t));
                    hipMalloc((void**)&deviceAln[gn], 2*seqLen*alignSize[rn] * sizeof(int8_t));
                    hipMalloc((void**)&deviceLen[gn], 2*alignSize[rn] * sizeof(int32_t));
                    hipMalloc((void**)&deviceAlnLen[gn], alignSize[rn] * sizeof(int32_t));
                    // hipMalloc((void**)&deviceSeqInfo[gn], 5 * sizeof(int32_t));
                    // error = hipGetLastError(); printf("CUDA error Alloc: %s\n",hipGetErrorString(error)); 
                }
                
                hipMemcpy(deviceFreq[gn], hostFreq[rn], 12*seqLen * alignSize[rn] * sizeof(uint16_t), hipMemcpyHostToDevice);
                // error = hipGetLastError(); printf("CUDA error Freq: %s\n",hipGetErrorString(error)); 
                hipMemcpy(deviceAln[gn], hostAln[rn], 2*seqLen * alignSize[rn] * sizeof(int8_t), hipMemcpyHostToDevice);
                // error = hipGetLastError(); printf("CUDA error Aln: %s\n",hipGetErrorString(error)); 
                hipMemcpy(deviceLen[gn], hostLen[rn], 2*alignSize[rn] * sizeof(int32_t), hipMemcpyHostToDevice);
                // error = hipGetLastError(); printf("CUDA error Len: %s\n",hipGetErrorString(error)); 
                hipMemcpy(deviceAlnLen[gn], hostAlnLen[rn], alignSize[rn] * sizeof(int32_t), hipMemcpyHostToDevice);
                // error = hipGetLastError(); printf("CUDA error AlnLen: %s\n",hipGetErrorString(error)); 
                hipMemcpy(deviceSeqInfo[gn], hostSeqInfo[rn], 5 * sizeof(int32_t), hipMemcpyHostToDevice);
                // error = hipGetLastError(); printf("CUDA error SeqInfo: %s\n",hipGetErrorString(error)); 
                // hipMemcpy(deviceParam[gn], hostParam[rn], 7 * sizeof(paramType), hipMemcpyHostToDevice);
                // error = hipGetLastError(); printf("CUDA error Param: %s\n",hipGetErrorString(error)); 
                std::string berr = hipGetErrorString(hipGetLastError());
                if (berr != "no error") printf("ERROR: Before kernel %s!\n", berr.c_str());
                alignGrpToGrp_talco<<<numBlocks, blockSize>>>(
                    deviceFreq[gn],
                    deviceAln[gn], 
                    deviceLen[gn],
                    deviceAlnLen[gn],
                    deviceSeqInfo[gn], 
                    deviceParam[gn]
                );
                hipDeviceSynchronize();
                std::string aerr = hipGetErrorString(hipGetLastError());
                if (aerr != "no error") printf("ERROR: After kernel %s!\n", aerr.c_str());
                hipMemcpy(hostAln[rn], deviceAln[gn], 2*seqLen * alignSize[rn] * sizeof(int8_t), hipMemcpyDeviceToHost);
                // error = hipGetLastError(); printf("CUDA error rAln: %s\n",hipGetErrorString(error)); 
                hipMemcpy(hostAlnLen[rn], deviceAlnLen[gn], alignSize[rn] * sizeof(int32_t), hipMemcpyDeviceToHost);
                hipDeviceSynchronize();  
                int maxAlnLen = 0;
                for (int k = 0; k <  alignSize[rn]; ++k) {
                    if (hostAlnLen[rn][k] > maxAlnLen) maxAlnLen = hostAlnLen[rn][k];
                }
                std::cout << "GPU: " << gn << " Rn: " << rn << " maxLen: " << maxAlnLen << "\n";
                
            }
        }
    });

    // free memory  
    for (int gn = 0; gn < gpuNum; ++gn) {
        hipSetDevice(gn);
        hipFree(deviceFreq[gn]);
        hipFree(deviceAlnLen[gn]);
        hipFree(deviceLen[gn]);
        hipFree(deviceAln[gn]);
        hipFree(deviceParam[gn]);
        hipFree(deviceSeqInfo[gn]);
        hipDeviceSynchronize();  
    }

    auto kernelEnd = std::chrono::high_resolution_clock::now();
    std::chrono::nanoseconds kernelTime = kernelEnd - kernelStart;
    int totalPairs = 0;
    for (int gn = 0; gn < roundGPU; ++gn) totalPairs += alignSize[gn];
    std::cout << "GPU KernelTime "<< kernelTime.count() / 1000000<< " ms\n";

    auto reAlnStart = std::chrono::high_resolution_clock::now();
    
    int maxAlnLen = 0;
    for (int gn = 0; gn < roundGPU; ++gn) {
       for (int k = 0; k <  alignSize[gn]; ++k) {
            if (hostAlnLen[gn][k] > maxAlnLen) maxAlnLen = hostAlnLen[gn][k];
        }
    }
    util->memCheck(maxAlnLen);
        
    for (int gn = 0; gn < roundGPU; ++gn) {
        if (alignSize[gn] == 0) break;
        tbb::parallel_for(tbb::blocked_range<int>(0, alignSize[gn]), [&](tbb::blocked_range<int> range) {
            // for (int k = 0; k < alignSize[gn]; ++k) {
            for (int k = range.begin(); k < range.end(); ++k) {
                // std::vector<std::string> alignment;
                int32_t refNum = seqIdx[gn][k].second - seqIdx[gn][k].first;
                int32_t qryNum = (k !=  alignSize[gn]-1) ? seqIdx[gn][k+1].first - seqIdx[gn][k].second : seqNum[gn] - seqIdx[gn][k].second;
                int32_t refStart = seqIdx[gn][k].first;
                int32_t qryStart = seqIdx[gn][k].second;
                int32_t nIdx = k + gn*numBlocks;
                if (hostAlnLen[gn][k] <= 0) {
                    int32_t refLen = util->seqsLen[nodes[nIdx].first->identifier];
                    int32_t qryLen = util->seqsLen[nodes[nIdx].second->identifier];
                    std::vector<int8_t> aln;
                    alignGrpToGrp_traditional
                    (
                        freq[gn][k],
                        seqLen,
                        refLen,
                        qryLen,
                        param,
                        aln
                    );
                    int32_t alnLen = aln.size();
                    util->memCheck(alnLen);
                    std::reverse(aln.begin(), aln.end());
                    for (auto sIdx: tree->allNodes[nodes[nIdx].first->identifier]->msaIdx) {
                        int storeFrom = util->seqsStorage[sIdx];
                        int storeTo = 1 - util->seqsStorage[sIdx];
                        int orgIdx = 0;
                        for (int j = 0; j < aln.size(); ++j) {
                            if ((aln[j] & 0xFFFF) == 0 || (aln[j] & 0xFFFF) == 2) {
                                util->seqBuf[storeTo][sIdx][j] = util->seqBuf[storeFrom][sIdx][orgIdx];
                                orgIdx++;
                            }
                            else {
                                util->seqBuf[storeTo][sIdx][j] = '-';
                            }
                        }
                        util->seqsLen[nodes[nIdx].first->identifier] = aln.size();
                        util->changeStorage(sIdx);
                    }
                    for (auto sIdx: tree->allNodes[nodes[nIdx].second->identifier]->msaIdx) {
                        int storeFrom = util->seqsStorage[sIdx];
                        int storeTo = 1 - util->seqsStorage[sIdx];
                        int orgIdx = 0;
                        for (int j = 0; j < aln.size(); ++j) {
                            if ((aln[j] & 0xFFFF) == 0 || (aln[j] & 0xFFFF) == 1) {
                                util->seqBuf[storeTo][sIdx][j] = util->seqBuf[storeFrom][sIdx][orgIdx];
                                orgIdx++;
                            }
                            else {
                                util->seqBuf[storeTo][sIdx][j] = '-';
                            }
                        }
                        util->seqsLen[nodes[nIdx].second->identifier] = aln.size();
                        util->changeStorage(sIdx);
                    }
                    printf("CPU fallback (traditional global alignment) on No. %d (%s), Alignment Length: %d\n", k, tree->allNodes[nodes[nIdx].first->identifier]->identifier.c_str(), aln.size());
                    // printf("CPU fallback on No. %d (%s), Alignment Length: %d\n", k, tree->allNodes[nodes[nIdx].first->identifier]->identifier.c_str(), aln.size());
                }
                // else if (hostAlnLen[gn][k] <= 0) {
                //     std::vector<int8_t> aln;
                //     std::vector<std::vector<int>> freqRef;
                //     std::vector<std::vector<int>> freqQry;
                //     int32_t refLen = util->seqsLen[nodes[nIdx].first->identifier];
                //     int32_t qryLen = util->seqsLen[nodes[nIdx].second->identifier];
                    
                //     for (int r = 0; r < refLen; r++) {
                //         std::vector<int> temp;
                //         for (int f = 0; f < 6; ++f) temp.push_back(freq[gn][k][6*r+f]);
                //         freqRef.push_back(temp);
                //     }
                //     for (int q = 0; q < qryLen; q++) {
                //         std::vector<int> temp;
                //         for (int f = 0; f < 6; ++f) temp.push_back(freq[gn][k][6*(seqLen+q)+f]);
                //         freqQry.push_back(temp);
                //     }
                //     Talco_xdrop::Params talco_params(param.match, param.mismatch, param.gapOpen, param.gapExtend, 1000, param.marker);
                //     Talco_xdrop::Align_freq (
                //         talco_params,
                //         freqRef,
                //         freqQry,
                //         aln
                //     );
                //     util->memCheck(aln.size());
                //     for (auto sIdx: tree->allNodes[nodes[nIdx].first->identifier]->msaIdx) {
                //         int64_t start = sIdx*util->memLen;
                //         int storeFrom = util->seqsStorage[sIdx];
                //         int storeTo = 1 - util->seqsStorage[sIdx];
                //         int orgIdx = 0;
                //         for (int j = 0; j < aln.size(); ++j) {
                //             if ((aln[j] & 0xFFFF) == 0 || (aln[j] & 0xFFFF) == 2) {
                //                 util->seqBuf[storeTo][start+j] = util->seqBuf[storeFrom][start+orgIdx];
                //                 orgIdx++;
                //             }
                //             else {
                //                 util->seqBuf[storeTo][start+j] = '-';
                //             }
                //         }
                //         util->seqsLen[nodes[nIdx].first->identifier] = aln.size();
                //         util->changeStorage(sIdx);
                //     }
                //     for (auto sIdx: tree->allNodes[nodes[nIdx].second->identifier]->msaIdx) {
                //         int64_t start = sIdx*util->memLen;
                //         int storeFrom = util->seqsStorage[sIdx];
                //         int storeTo = 1 - util->seqsStorage[sIdx];
                //         int orgIdx = 0;
                //         for (int j = 0; j < aln.size(); ++j) {
                //             if ((aln[j] & 0xFFFF) == 0 || (aln[j] & 0xFFFF) == 1) {
                //                 util->seqBuf[storeTo][start+j] = util->seqBuf[storeFrom][start+orgIdx];
                //                 orgIdx++;
                //             }
                //             else {
                //                 util->seqBuf[storeTo][start+j] = '-';
                //             }
                //         }
                //         util->seqsLen[nodes[nIdx].second->identifier] = aln.size();
                //         util->changeStorage(sIdx);
                //     }
                //     printf("CPU fallback (TALCO-Xdrop) on No. %d (%s), Alignment Length: %d\n", k, tree->allNodes[nodes[nIdx].first->identifier]->identifier.c_str(), aln.size());
                // }
                else {
                    for (auto sIdx: tree->allNodes[nodes[nIdx].first->identifier]->msaIdx) {
                        int orgIdx = 0;
                        int storeFrom = util->seqsStorage[sIdx];
                        int storeTo = 1 - util->seqsStorage[sIdx];
                        for (int j = 0; j < hostAlnLen[gn][k]; ++j) {
                            if ((hostAln[gn][k*2*seqLen+j] & 0xFFFF) == 0 || (hostAln[gn][k*2*seqLen+j] & 0xFFFF) == 2) {
                                util->seqBuf[storeTo][sIdx][j] = util->seqBuf[storeFrom][sIdx][orgIdx];
                                orgIdx++;
                            }
                            else {
                                util->seqBuf[storeTo][sIdx][j] = '-';
                            }
                        }
                        util->seqsLen[nodes[nIdx].first->identifier] = hostAlnLen[gn][k];
                        util->changeStorage(sIdx);
                    }
                    for (auto sIdx: tree->allNodes[nodes[nIdx].second->identifier]->msaIdx) {
                        int storeFrom = util->seqsStorage[sIdx];
                        int storeTo = 1 - util->seqsStorage[sIdx];
                        int orgIdx = 0;
                        for (int j = 0; j < hostAlnLen[gn][k]; ++j) {
                            if ((hostAln[gn][k*2*seqLen+j] & 0xFFFF) == 0 || (hostAln[gn][k*2*seqLen+j] & 0xFFFF) == 1) {
                                util->seqBuf[storeTo][sIdx][j] = util->seqBuf[storeFrom][sIdx][orgIdx];
                                orgIdx++;
                            }
                            else {
                                util->seqBuf[storeTo][sIdx][j] = '-';
                            }
                        }
                        util->seqsLen[nodes[nIdx].second->identifier] = hostAlnLen[gn][k];
                        util->changeStorage(sIdx);
                    }
                }
                // std::cout << "LenB : " << nodes[nIdx].first->identifier << '(' << tree->allNodes[nodes[nIdx].first->identifier]->msaIdx.size() << ')'
                //                       << nodes[nIdx].second->identifier << '(' << tree->allNodes[nodes[nIdx].second->identifier]->msaIdx.size() << ")\n";
                for (auto q: tree->allNodes[nodes[nIdx].second->identifier]->msaIdx) 
                    tree->allNodes[nodes[nIdx].first->identifier]->msaIdx.push_back(q);
            }  
        });
        for (int i = 0; i < alignSize[gn]; ++i) delete [] freq[gn][i];
    } 
    auto reAlnEnd = std::chrono::high_resolution_clock::now();
    std::chrono::nanoseconds reAlnTime = reAlnEnd - kernelEnd;
    printf("Alignment Time: %d us\n", reAlnTime.count() / 1000);

    

    for (int rn = 0; rn < roundGPU; ++rn) {
        free(hostFreq[rn]);
        free(hostAlnLen[rn]);
        free(hostLen[rn]);
        free(hostAln[rn]);
        free(hostSeqInfo[rn]);
    }  
    free(hostParam);

    delete [] alignSize;
    delete [] seqNum;
    delete [] deviceFreq;
    delete [] deviceAlnLen;
    delete [] deviceAln;
    delete [] deviceParam;
    delete [] deviceSeqInfo;
    delete [] hostFreq;
    delete [] hostAlnLen;
    delete [] hostAln;
    delete [] hostLen;
    // delete [] hostParam;
    delete [] hostSeqInfo;
    return;
}
*/


void msaPostOrderTraversal_multigpu(Tree* tree, std::vector<std::pair<Node*, Node*>>& nodes, msa::utility* util, Params& param)
{

    // auto freqStart = std::chrono::high_resolution_clock::now();
    for (auto n_pair: nodes) {
        auto n = std::make_pair(tree->allNodes[n_pair.first->identifier], tree->allNodes[n_pair.second->identifier]);
        if (n.first->children.size()==0) {
            tree->allNodes[n.first->identifier]->msaIdx.push_back(util->seqsIdx[n.first->identifier]);
        }
        else {
            if (tree->allNodes[n.first->identifier]->msaIdx.size() == 0) {
                Node* node = tree->allNodes[n.first->identifier];
                int grpID = node->grpID;
                for (int childIndex=0; childIndex<node->children.size(); childIndex++) {
                    if ((node->children[childIndex]->grpID == -1 || node->children[childIndex]->grpID == grpID) && (node->children[childIndex]->identifier != n.second->identifier)) {
                        if (node->children[childIndex]->msaIdx.size() == 0) tree->allNodes[node->children[childIndex]->identifier]->msaIdx.push_back(util->seqsIdx[node->children[childIndex]->identifier]);
                        tree->allNodes[n.first->identifier]->msaIdx = node->children[childIndex]->msaIdx;
                        util->seqsLen[n.first->identifier] = util->seqsLen[node->children[childIndex]->identifier];
                        break;
                    }
                }
            }
        }
        if (n.second->children.size()==0) {
            tree->allNodes[n.second->identifier]->msaIdx.push_back(util->seqsIdx[n.second->identifier]);
        }
        else {
            if (tree->allNodes[n.second->identifier]->msaIdx.size() == 0) {
                Node* node = tree->allNodes[n.second->identifier];
                int grpID = node->grpID;
                for (int childIndex=0; childIndex<node->children.size(); childIndex++) {
                    if ((node->children[childIndex]->grpID == -1 || node->children[childIndex]->grpID == grpID) && (node->children[childIndex]->identifier != n.first->identifier)) {
                        if (node->children[childIndex]->msaIdx.size() == 0) tree->allNodes[node->children[childIndex]->identifier]->msaIdx.push_back(util->seqsIdx[node->children[childIndex]->identifier]);
                        tree->allNodes[n.second->identifier]->msaIdx = node->children[childIndex]->msaIdx;
                        util->seqsLen[n.second->identifier] = util->seqsLen[node->children[childIndex]->identifier];
                        break;
                    }
                }
            }
        }
    }

    int numBlocks = 1024; 
    int blockSize = THREAD_NUM;
    int gpuNum = util->gpuNum;
    // hipGetDeviceCount(&gpuNum); // number of CUDA devices
    
    // get maximum sequence/profile length 
    int32_t seqLen = util->memLen;
    int roundGPU = nodes.size() / numBlocks + 1;
    if (nodes.size()%numBlocks == 0) roundGPU -= 1;
    if (roundGPU < gpuNum) gpuNum = roundGPU;
    
    paramType* hostParam = (paramType*)malloc(28 * sizeof(paramType)); 

    if (!param.userDefine) {
        for (int i = 0; i < 5; ++i) {
            for (int j = 0; j < 5; ++j) {
                if (i == 5 || j == 5)          hostParam[i*5+j] = 0;
                else if (i == j)               hostParam[i*5+j] = param.match;
                else if (i-j == 2 || j-i == 2) hostParam[i*5+j] = param.trans;
                else                           hostParam[i*5+j] = param.mismatch;
            }
        }
        hostParam[25] = param.gapOpen;
        hostParam[26] = param.gapExtend;
        hostParam[27] = param.xdrop;
    }
    else {
        for (int i = 0; i < 5; ++i) for (int j = 0; j < 5; ++j) hostParam[i*5+j] = param.userMatrix[i][j];
        hostParam[25] = param.userGapOpen;
        hostParam[26] = param.userGapExtend;
        hostParam[27] = param.xdrop;
    }
    

    std::vector<std::vector<std::pair<int32_t, int32_t>>> seqIdx;
    // allocate memory on host and device
    uint16_t** hostFreq = new uint16_t* [gpuNum];
    int8_t**   hostAln = new int8_t* [gpuNum];
    int32_t**  hostLen = new int32_t* [gpuNum];
    int32_t**  hostAlnLen = new int32_t* [gpuNum];
    int32_t**  hostSeqInfo = new int32_t* [gpuNum];

    uint16_t** deviceFreq = new uint16_t* [gpuNum];
    int8_t**   deviceAln = new int8_t* [gpuNum];
    int32_t**  deviceLen = new int32_t* [gpuNum];
    int32_t**  deviceAlnLen = new int32_t* [gpuNum];
    int32_t**  deviceSeqInfo = new int32_t* [gpuNum];
    
    paramType**  deviceParam = new paramType* [gpuNum];

    std::atomic<int> nowRound;
    nowRound.store(0);
    // nowRound.store(roundGPU-1);


    int maxThreads = tbb::this_task_arena::max_concurrency();
    
    // int ThreadsPerGPU = maxThreads / gpuNum;
    bool* cpuFallback = new bool[nodes.size()];
    for (int i = 0; i < nodes.size(); ++i) cpuFallback[i] = false;

    tbb::parallel_for(tbb::blocked_range<int>(0, gpuNum), [&](tbb::blocked_range<int> range){ 
        for (int gn = range.begin(); gn < range.end(); ++gn) {
            hostFreq[gn] = (uint16_t*)malloc(12 * seqLen * numBlocks * sizeof(uint16_t));
            hostAln[gn] = (int8_t*)malloc(    2 * seqLen * numBlocks * sizeof(int8_t));
            hostLen[gn] = (int32_t*)malloc(   2 *          numBlocks * sizeof(int32_t));
            hostAlnLen[gn] = (int32_t*)malloc(             numBlocks * sizeof(int32_t));
            hostSeqInfo[gn] = (int32_t*)malloc(5                     * sizeof(int32_t));
            
            hipSetDevice(gn);
            // hipError_t error;
            hipMalloc((void**)&deviceFreq[gn],  12 * seqLen * numBlocks * sizeof(uint16_t));
            hipMalloc((void**)&deviceAln[gn],    2 * seqLen * numBlocks * sizeof(int8_t));
            hipMalloc((void**)&deviceLen[gn],    2 *          numBlocks * sizeof(int32_t));
            hipMalloc((void**)&deviceAlnLen[gn],              numBlocks * sizeof(int32_t));
            hipMalloc((void**)&deviceSeqInfo[gn], 5 * sizeof(int32_t));
            hipMalloc((void**)&deviceParam[gn],  28 * sizeof(paramType));

            hipMemcpy(deviceParam[gn], hostParam, 28 * sizeof(paramType), hipMemcpyHostToDevice);
            // error = hipGetLastError(); printf("CUDA error Malloc: %s\n",hipGetErrorString(error)); 
            std::vector<std::pair<int, int>> seqIdx;
            
            
            while (nowRound < roundGPU) {
            // while (nowRound >= 0) {
                int rn = nowRound.fetch_add(1);
                int alnPairs = (nodes.size() - rn*numBlocks > numBlocks) ? numBlocks : nodes.size() - rn*numBlocks;
                int seqNum = 0;
                // std::cout << "GPU: " << gn << " Rn: " << rn << " Pairs: " << alnPairs << '\n';

                // Initailize 
                for (int n = 0; n < 12*seqLen * numBlocks; ++n) hostFreq[gn][n] = 0;
                for (int n = 0; n <  2*seqLen * numBlocks; ++n) hostAln[gn][n] = 0;
                for (int n = 0; n <  2*         numBlocks; ++n) hostLen[gn][n] = 0;
                for (int n = 0; n <             numBlocks; ++n) hostAlnLen[gn][n] = 0;
                seqIdx.clear();

                // Calculate Frequency
                // tbb::task_arena ta {maxThreads - gpuNum};
                // tbb::task_group tg;
                // tg.run ([&] () {
                // tbb::this_task_arena::isolate ([&] () {
                // std::cout << "SSS\n";
                // tbb::parallel_for(tbb::blocked_range<int>(0, alnPairs), [&](tbb::blocked_range<int> aln_range) { 
                // for (int n = aln_range.begin(); n < aln_range.end(); ++n) {
                
                for (int n = 0; n < alnPairs; ++n) {
                    int32_t nIdx = n + rn*numBlocks;
                    int32_t qryIdx = 0;
                    int32_t refIdx = 0;
                    int32_t refLen = util->seqsLen[nodes[nIdx].first->identifier];
                    int32_t qryLen = util->seqsLen[nodes[nIdx].second->identifier];
                    refIdx = seqNum;
                    for (auto sIdx: tree->allNodes[nodes[nIdx].first->identifier]->msaIdx) { 
                        int storage = util->seqsStorage[sIdx];
                        int maxLen = max(refLen, qryLen);
                        tbb::this_task_arena::isolate( [&]{
                        tbb::parallel_for(tbb::blocked_range<int>(0, refLen), [&](tbb::blocked_range<int> r) {
                        for (int s = r.begin(); s < r.end(); ++s) {
                        // for (int s = 0; s < refLen; ++s) {
                            if      (util->alnStorage[storage][sIdx][s] == 'A' || util->alnStorage[storage][sIdx][s] == 'a') hostFreq[gn][12*seqLen*n+6*s+0]+=1;
                            else if (util->alnStorage[storage][sIdx][s] == 'C' || util->alnStorage[storage][sIdx][s] == 'c') hostFreq[gn][12*seqLen*n+6*s+1]+=1;
                            else if (util->alnStorage[storage][sIdx][s] == 'G' || util->alnStorage[storage][sIdx][s] == 'g') hostFreq[gn][12*seqLen*n+6*s+2]+=1;
                            else if (util->alnStorage[storage][sIdx][s] == 'T' || util->alnStorage[storage][sIdx][s] == 't' ||
                                     util->alnStorage[storage][sIdx][s] == 'U' || util->alnStorage[storage][sIdx][s] == 'u') hostFreq[gn][12*seqLen*n+6*s+3]+=1;
                            else if (util->alnStorage[storage][sIdx][s] == 'N' || util->alnStorage[storage][sIdx][s] == 'n') hostFreq[gn][12*seqLen*n+6*s+4]+=1;
                            else                                                                                             hostFreq[gn][12*seqLen*n+6*s+5]+=1;
                        }
                        });
                        });
                        seqNum += 1;
                    }
                    qryIdx = seqNum;
                    for (auto sIdx: tree->allNodes[nodes[nIdx].second->identifier]->msaIdx) { 
                    int storage = util->seqsStorage[sIdx];
                    tbb::this_task_arena::isolate( [&]{
                    tbb::parallel_for(tbb::blocked_range<int>(0, qryLen), [&](tbb::blocked_range<int> r) {
                    for (int s = r.begin(); s < r.end(); ++s) {
                    // for (int s = 0; s < qryLen; ++s) {
                        if      (util->alnStorage[storage][sIdx][s] == 'A' || util->alnStorage[storage][sIdx][s] == 'a') hostFreq[gn][12*seqLen*n+6*(seqLen+s)+0]+=1;
                        else if (util->alnStorage[storage][sIdx][s] == 'C' || util->alnStorage[storage][sIdx][s] == 'c') hostFreq[gn][12*seqLen*n+6*(seqLen+s)+1]+=1;
                        else if (util->alnStorage[storage][sIdx][s] == 'G' || util->alnStorage[storage][sIdx][s] == 'g') hostFreq[gn][12*seqLen*n+6*(seqLen+s)+2]+=1;
                        else if (util->alnStorage[storage][sIdx][s] == 'T' || util->alnStorage[storage][sIdx][s] == 't' ||
                                 util->alnStorage[storage][sIdx][s] == 'U' || util->alnStorage[storage][sIdx][s] == 'u') hostFreq[gn][12*seqLen*n+6*(seqLen+s)+3]+=1;
                        else if (util->alnStorage[storage][sIdx][s] == 'N' || util->alnStorage[storage][sIdx][s] == 'n') hostFreq[gn][12*seqLen*n+6*(seqLen+s)+4]+=1;
                        else                                                                                             hostFreq[gn][12*seqLen*n+6*(seqLen+s)+5]+=1;
                    }
                    });
                    });
                    seqNum += 1;
                }
                    hostLen[gn][2*n] = refLen; hostLen[gn][2*n+1] = qryLen;
                    seqIdx.push_back(std::make_pair(refIdx, qryIdx));
                }
                
                hostSeqInfo[gn][0] = seqLen;
                hostSeqInfo[gn][1] = seqNum;
                hostSeqInfo[gn][2] = alnPairs;
                hostSeqInfo[gn][3] = numBlocks;
                hostSeqInfo[gn][4] = param.userDefine;
        
                hipMemcpy(deviceFreq[gn], hostFreq[gn], 12*seqLen * numBlocks * sizeof(uint16_t), hipMemcpyHostToDevice);
                hipMemcpy(deviceAln[gn], hostAln[gn], 2*seqLen * numBlocks * sizeof(int8_t), hipMemcpyHostToDevice);
                hipMemcpy(deviceLen[gn], hostLen[gn], 2*numBlocks * sizeof(int32_t), hipMemcpyHostToDevice);
                hipMemcpy(deviceAlnLen[gn], hostAlnLen[gn], numBlocks * sizeof(int32_t), hipMemcpyHostToDevice);
                hipMemcpy(deviceSeqInfo[gn], hostSeqInfo[gn], 5 * sizeof(int32_t), hipMemcpyHostToDevice);
                
                std::string berr = hipGetErrorString(hipGetLastError());
                if (berr != "no error") printf("ERROR: Before kernel %s!\n", berr.c_str());
                alignGrpToGrp_talco<<<numBlocks, blockSize>>>(
                    deviceFreq[gn],
                    deviceAln[gn], 
                    deviceLen[gn],
                    deviceAlnLen[gn],
                    deviceSeqInfo[gn], 
                    deviceParam[gn]
                );
                hipDeviceSynchronize();
                std::string aerr = hipGetErrorString(hipGetLastError());
                if (aerr != "no error") printf("ERROR: After kernel %s!\n", aerr.c_str());
                
                hipMemcpy(hostAln[gn], deviceAln[gn], 2*seqLen * numBlocks * sizeof(int8_t), hipMemcpyDeviceToHost);
                hipMemcpy(hostAlnLen[gn], deviceAlnLen[gn], numBlocks * sizeof(int32_t), hipMemcpyDeviceToHost);
                hipDeviceSynchronize();
                int maxAlnLen = 0;
                for (int n = 0; n <  alnPairs; ++n) {
                    if (hostAlnLen[gn][n] > maxAlnLen) maxAlnLen = hostAlnLen[gn][n];
                }
                util->memCheck(maxAlnLen);
                if (rn % 10 == 0 && rn > 0) std::cout << rn*numBlocks << " pairs have been processed.\n";
                
                tbb::this_task_arena::isolate( [&]{
                tbb::parallel_for(tbb::blocked_range<int>(0, alnPairs), [&](tbb::blocked_range<int> range) {
                for (int n = range.begin(); n < range.end(); ++n) {
                // for (int n = 0; n < alnPairs; ++n) {
                    
                    int32_t refNum = seqIdx[n].second - seqIdx[n].first;
                    int32_t qryNum = (n !=  alnPairs-1) ? seqIdx[n+1].first - seqIdx[n].second : seqNum - seqIdx[n].second;
                    int32_t nIdx = n + rn*numBlocks;

                    // if (nIdx % 400 == 399) {
                    if (hostAlnLen[gn][n] <= 0) {
                        cpuFallback[nIdx] = true;
                        // int32_t refLen = util->seqsLen[nodes[nIdx].first->identifier];
                        // int32_t qryLen = util->seqsLen[nodes[nIdx].second->identifier];
                        // uint16_t *freq = new uint16_t[12*seqLen]; 
                        // for (int i = 0; i < 12*seqLen; ++i) freq[i] = hostFreq[gn][12*seqLen*n+i];
                        // std::vector<int8_t> aln;
                        // alignGrpToGrp_traditional (
                        //     freq,
                        //     seqLen,
                        //     refLen,
                        //     qryLen,
                        //     param,
                        //     aln
                        // );
                        // delete [] freq;
                        // int32_t alnLen = aln.size();
                        // util->memCheck(alnLen);
                        // std::reverse(aln.begin(), aln.end());
                        // for (auto sIdx: tree->allNodes[nodes[nIdx].first->identifier]->msaIdx) {
                        //     int storeFrom = util->seqsStorage[sIdx];
                        //     int storeTo = 1 - util->seqsStorage[sIdx];
                        //     int orgIdx = 0;
                        //     for (int j = 0; j < aln.size(); ++j) {
                        //         if ((aln[j] & 0xFFFF) == 0 || (aln[j] & 0xFFFF) == 2) {
                        //             util->alnStorage[storeTo][sIdx][j] = util->alnStorage[storeFrom][sIdx][orgIdx];
                        //             orgIdx++;
                        //         }
                        //         else {
                        //             util->alnStorage[storeTo][sIdx][j] = '-';
                        //         }
                        //     }
                        //     util->seqsLen[nodes[nIdx].first->identifier] = aln.size();
                        //     util->changeStorage(sIdx);
                        // }
                        // for (auto sIdx: tree->allNodes[nodes[nIdx].second->identifier]->msaIdx) {
                        //     int storeFrom = util->seqsStorage[sIdx];
                        //     int storeTo = 1 - util->seqsStorage[sIdx];
                        //     int orgIdx = 0;
                        //     for (int j = 0; j < aln.size(); ++j) {
                        //         if ((aln[j] & 0xFFFF) == 0 || (aln[j] & 0xFFFF) == 1) {
                        //             util->alnStorage[storeTo][sIdx][j] = util->alnStorage[storeFrom][sIdx][orgIdx];
                        //             orgIdx++;
                        //         }
                        //         else {
                        //             util->alnStorage[storeTo][sIdx][j] = '-';
                        //         }
                        //     }
                        //     util->seqsLen[nodes[nIdx].second->identifier] = aln.size();
                        //     util->changeStorage(sIdx);
                        // }
                        // std::cout << "CPU fallback (traditional global alignment) on No. " << n << " (" << tree->allNodes[nodes[nIdx].first->identifier]->identifier << ")\n";
                    }
                    else {
                        for (auto sIdx: tree->allNodes[nodes[nIdx].first->identifier]->msaIdx) {
                            int orgIdx = 0;
                            int storeFrom = util->seqsStorage[sIdx];
                            int storeTo = 1 - util->seqsStorage[sIdx];
                            for (int j = 0; j < hostAlnLen[gn][n]; ++j) {
                                if ((hostAln[gn][n*2*seqLen+j] & 0xFFFF) == 0 || (hostAln[gn][n*2*seqLen+j] & 0xFFFF) == 2) {
                                    util->alnStorage[storeTo][sIdx][j] = util->alnStorage[storeFrom][sIdx][orgIdx];
                                    orgIdx++;
                                }
                                else {
                                    util->alnStorage[storeTo][sIdx][j] = '-';
                                }
                            }
                            util->seqsLen[nodes[nIdx].first->identifier] = hostAlnLen[gn][n];
                            util->changeStorage(sIdx);
                        }
                        for (auto sIdx: tree->allNodes[nodes[nIdx].second->identifier]->msaIdx) {
                            int storeFrom = util->seqsStorage[sIdx];
                            int storeTo = 1 - util->seqsStorage[sIdx];
                            int orgIdx = 0;
                            for (int j = 0; j < hostAlnLen[gn][n]; ++j) {
                                if ((hostAln[gn][n*2*seqLen+j] & 0xFFFF) == 0 || (hostAln[gn][n*2*seqLen+j] & 0xFFFF) == 1) {
                                    util->alnStorage[storeTo][sIdx][j] = util->alnStorage[storeFrom][sIdx][orgIdx];
                                    orgIdx++;
                                }
                                else {
                                    util->alnStorage[storeTo][sIdx][j] = '-';
                                }
                            }
                            util->seqsLen[nodes[nIdx].second->identifier] = hostAlnLen[gn][n];
                            util->changeStorage(sIdx);
                        }
                        for (auto q: tree->allNodes[nodes[nIdx].second->identifier]->msaIdx) {
                            tree->allNodes[nodes[nIdx].first->identifier]->msaIdx.push_back(q);
                        }
                        tree->allNodes[nodes[nIdx].second->identifier]->msaIdx.clear();
                    }
                    
                }
                });
                });
                
            }  

            
        }
    });
    
    // free memory  
    free(hostParam);
    for (int gn = 0; gn < gpuNum; ++gn) {
        hipSetDevice(gn);
        hipFree(deviceFreq[gn]);
        hipFree(deviceAln[gn]);
        hipFree(deviceLen[gn]);
        hipFree(deviceAlnLen[gn]);
        hipFree(deviceSeqInfo[gn]);
        hipFree(deviceParam[gn]);
        hipDeviceSynchronize();  
        free(hostFreq[gn]);
        free(hostAln[gn]);
        free(hostLen[gn]);
        free(hostAlnLen[gn]);
        free(hostSeqInfo[gn]);
    }
    
    delete [] deviceFreq;
    delete [] deviceAlnLen;
    delete [] deviceAln;
    delete [] deviceParam;
    delete [] deviceSeqInfo;
    delete [] deviceLen;
    delete [] hostFreq;
    delete [] hostAlnLen;
    delete [] hostLen;
    delete [] hostAln;
    delete [] hostSeqInfo;
    
    
    // CPU Fallback
    std::vector<int> fallbackPairs;
    for (int i = 0; i < nodes.size(); ++i) if (cpuFallback[i]) fallbackPairs.push_back(i);
    delete [] cpuFallback;
    if (fallbackPairs.size() > 0) std::cout << "CPU Fallback. Num of pairs: " << fallbackPairs.size() << '\n';
    else return;
    tbb::parallel_for(tbb::blocked_range<int>(0, fallbackPairs.size()), [&](tbb::blocked_range<int> range) {
    for (int n = range.begin(); n < range.end(); ++n) {
                
    // for (auto nIdx: fallbackPairs) {
        int nIdx = fallbackPairs[n];
        uint16_t *freq = new uint16_t[12*seqLen]; 
        for (int i = 0; i < 12*seqLen; ++i) freq[i] = 0;
        int32_t qryIdx = 0;
        int32_t refIdx = 0;
        int32_t refLen = util->seqsLen[nodes[nIdx].first->identifier];
        int32_t qryLen = util->seqsLen[nodes[nIdx].second->identifier];
        for (auto sIdx: tree->allNodes[nodes[nIdx].first->identifier]->msaIdx) { 
            int storage = util->seqsStorage[sIdx];
            int maxLen = max(refLen, qryLen);
            // tbb::parallel_for(tbb::blocked_range<int>(0, refLen), [&](tbb::blocked_range<int> r) {
            // for (int s = r.begin(); s < r.end(); ++s) {
            for (int s = 0; s < refLen; ++s) {
                if      (util->alnStorage[storage][sIdx][s] == 'A' || util->alnStorage[storage][sIdx][s] == 'a') freq[6*s+0]+=1;
                else if (util->alnStorage[storage][sIdx][s] == 'C' || util->alnStorage[storage][sIdx][s] == 'c') freq[6*s+1]+=1;
                else if (util->alnStorage[storage][sIdx][s] == 'G' || util->alnStorage[storage][sIdx][s] == 'g') freq[6*s+2]+=1;
                else if (util->alnStorage[storage][sIdx][s] == 'T' || util->alnStorage[storage][sIdx][s] == 't' ||
                         util->alnStorage[storage][sIdx][s] == 'U' || util->alnStorage[storage][sIdx][s] == 'u') freq[6*s+3]+=1;
                else if (util->alnStorage[storage][sIdx][s] == 'N' || util->alnStorage[storage][sIdx][s] == 'n') freq[6*s+4]+=1;
                else                                                                                             freq[6*s+5]+=1;
            }
            // });
        }
        for (auto sIdx: tree->allNodes[nodes[nIdx].second->identifier]->msaIdx) { 
            int storage = util->seqsStorage[sIdx];
            // tbb::parallel_for(tbb::blocked_range<int>(0, qryLen), [&](tbb::blocked_range<int> r) {
            // for (int s = r.begin(); s < r.end(); ++s) {
            for (int s = 0; s < qryLen; ++s) {
                if      (util->alnStorage[storage][sIdx][s] == 'A' || util->alnStorage[storage][sIdx][s] == 'a') freq[6*(seqLen+s)+0]+=1;
                else if (util->alnStorage[storage][sIdx][s] == 'C' || util->alnStorage[storage][sIdx][s] == 'c') freq[6*(seqLen+s)+1]+=1;
                else if (util->alnStorage[storage][sIdx][s] == 'G' || util->alnStorage[storage][sIdx][s] == 'g') freq[6*(seqLen+s)+2]+=1;
                else if (util->alnStorage[storage][sIdx][s] == 'T' || util->alnStorage[storage][sIdx][s] == 't' ||
                         util->alnStorage[storage][sIdx][s] == 'U' || util->alnStorage[storage][sIdx][s] == 'u') freq[6*(seqLen+s)+3]+=1;
                else if (util->alnStorage[storage][sIdx][s] == 'N' || util->alnStorage[storage][sIdx][s] == 'n') freq[6*(seqLen+s)+4]+=1;
                else                                                                                             freq[6*(seqLen+s)+5]+=1;
            }
            // });
        }
        std::vector<int8_t> aln;
        alignGrpToGrp_traditional (
            freq,
            seqLen,
            refLen,
            qryLen,
            param,
            aln
        );
        delete [] freq;
        int32_t alnLen = aln.size();
        util->memCheck(alnLen);
        std::reverse(aln.begin(), aln.end());
        for (auto sIdx: tree->allNodes[nodes[nIdx].first->identifier]->msaIdx) {
            int storeFrom = util->seqsStorage[sIdx];
            int storeTo = 1 - util->seqsStorage[sIdx];
            int orgIdx = 0;
            for (int j = 0; j < aln.size(); ++j) {
                if ((aln[j] & 0xFFFF) == 0 || (aln[j] & 0xFFFF) == 2) {
                    util->alnStorage[storeTo][sIdx][j] = util->alnStorage[storeFrom][sIdx][orgIdx];
                    orgIdx++;
                }
                else {
                    util->alnStorage[storeTo][sIdx][j] = '-';
                }
            }
            util->seqsLen[nodes[nIdx].first->identifier] = aln.size();
            util->changeStorage(sIdx);
        }
        for (auto sIdx: tree->allNodes[nodes[nIdx].second->identifier]->msaIdx) {
            int storeFrom = util->seqsStorage[sIdx];
            int storeTo = 1 - util->seqsStorage[sIdx];
            int orgIdx = 0;
            for (int j = 0; j < aln.size(); ++j) {
                if ((aln[j] & 0xFFFF) == 0 || (aln[j] & 0xFFFF) == 1) {
                    util->alnStorage[storeTo][sIdx][j] = util->alnStorage[storeFrom][sIdx][orgIdx];
                    orgIdx++;
                }
                else {
                    util->alnStorage[storeTo][sIdx][j] = '-';
                }
            }
            util->seqsLen[nodes[nIdx].second->identifier] = aln.size();
            util->changeStorage(sIdx);
        }
        for (auto q: tree->allNodes[nodes[nIdx].second->identifier]->msaIdx) {
            tree->allNodes[nodes[nIdx].first->identifier]->msaIdx.push_back(q);
        }
        tree->allNodes[nodes[nIdx].second->identifier]->msaIdx.clear();
        printf("CPU fallback (traditional global alignment) on No. %d (%s), Alignment Length: %d\n", nIdx, tree->allNodes[nodes[nIdx].first->identifier]->identifier.c_str(), aln.size());
    }
    });
    
    
    
    return;
}


void getMsaHierachy(std::vector<std::pair<std::pair<Node*, Node*>, int>>& hier, std::stack<Node*> msaStack, int grpID, int mode) {
    
    int hierIdx = 0;
    while(!msaStack.empty()) {
        Node* node = msaStack.top();
        if (!(node->grpID==-1 || node->grpID==grpID)) {
            msaStack.pop();
            continue;
        };
        if (node->children.size()==0) {
            msaStack.pop();
            continue;
        }
        std::vector<Node*> children;
        for (auto ch: node->children) {
            if (ch->grpID == grpID) children.push_back(ch);
        }
        if (mode == 0) {
            if (children.empty()) {
                node->grpID = -2;
                msaStack.pop();
                continue;
            }
            else if (children.size() == 1 && node->parent != nullptr) {
                if (node->parent->grpID == grpID) {
                    for (int chIdx = 0; chIdx < node->parent->children.size(); ++chIdx) {
                        if (node->parent->children[chIdx]->identifier == node->identifier) {
                            node->parent->children[chIdx] = children[0];
                        }
                    }
                    msaStack.pop();
                    continue;
                }
                
            }
        }
        
        size_t childIndex = 0;
        for (childIndex=0; childIndex<node->children.size(); childIndex++) {
            if ((node->children[childIndex]->grpID == -1 || node->children[childIndex]->grpID == grpID))
            {
                break;
            }
        }
        // std::cout << node->identifier << '\n';
        if (childIndex == node->children.size() - 1) {
            if (mode == 1) {
                hier.push_back(std::make_pair(std::make_pair(node, node->children[childIndex]),hierIdx));
                ++hierIdx;
            }
            msaStack.pop();
            continue;
        }
        int childIndexStart = (mode == 0) ? childIndex+1 : childIndex;
        for (size_t i=childIndexStart; i<node->children.size(); i++)
        {
            if (!(node->children[i]->grpID == -1 || node->children[i]->grpID == grpID))
            {
                continue;
            }
            hier.push_back(std::make_pair(std::make_pair(node, node->children[i]),hierIdx));
            ++hierIdx;
        }
        msaStack.pop();
    }
    
    hierIdx = 0;
    std::stack<std::pair<Node*, int>> subroots; 
    // Node* tempRoot = hier[0].first.first->parent;
    Node* preNode = hier[0].first.first;
    Node* preNode_2 = hier[0].first.second;
    size_t prelevel = hier[0].first.first->level;
    hier[0].second = hierIdx;
    
    for (int k = 1; k < hier.size(); ++k) { 
        if (!subroots.empty()) {
            if (hier[k].first.first->identifier == subroots.top().first->identifier) {
                hierIdx = max(hierIdx+1, subroots.top().second);
                hier[k].second = hierIdx; 
                prelevel = hier[k].first.first->level;
                subroots.pop();
            }
            else {
                if (mode == 0) {
                    if (hier[k].first.first->level < prelevel || hier[k].first.first->identifier == preNode->identifier) {
                        hier[k].second = ++hierIdx;
                        prelevel = hier[k].first.first->level;
                    }
                    else {
                        Node* tempNode = preNode;
                        while(true) {
                            Node* parent = tempNode->parent;
                            int childrenCount = 0;
                            for (auto ch: parent->children) {
                                if (ch->grpID == grpID) ++childrenCount;
                            }
                            if (childrenCount > 1) break;
                            tempNode = parent;
                        }
                        // hierStack.push(std::make_pair(preNode->parent, (hierIdx+1)));
                        if (tempNode->parent->identifier != subroots.top().first->identifier) {
                            subroots.push(std::make_pair(tempNode->parent, (hierIdx+1)));
                        }
                        else {
                            if (hierIdx >= subroots.top().second) {
                                subroots.pop();
                                subroots.push(std::make_pair(tempNode->parent, (hierIdx+1)));
                            }
                        }
                        hier[k].second = 0;
                        hierIdx = 0;
                        prelevel = hier[k].first.first->level;
                    }
                }
                else {
                    if (hier[k].first.first->level < prelevel || (hier[k].first.first->level == prelevel && hier[k].first.first->identifier == preNode->identifier)) {
                        hier[k].second = ++hierIdx;
                        prelevel = hier[k].first.first->level;
                    }
                    else {
                        if (preNode->parent->identifier == subroots.top().first->identifier) {
                            subroots.top().second = max(hierIdx+1, subroots.top().second);
                        }
                        else {
                            subroots.push(std::make_pair(preNode->parent, (hierIdx+1)));
                        }
                        
                        hier[k].second = 0;
                        hierIdx = 0;
                        prelevel = hier[k].first.first->level;
                    }
                }
            }
        }
        else {
            if (hier[k].first.first->level < prelevel || hier[k].first.first->identifier == preNode->identifier) {
                hier[k].second = ++hierIdx;
                prelevel = hier[k].first.first->level;
            }
            else {
                Node* tempNode = preNode;
                while(true) {
                    Node* parent = tempNode->parent;
                    int childrenCount = 0;
                    for (auto ch: parent->children) {
                        if (ch->grpID == grpID) ++childrenCount;
                    }
                    if (childrenCount > 1) break;
                    tempNode = parent;
                }
                subroots.push(std::make_pair(tempNode->parent, (hierIdx+1)));
                        
                hier[k].second = 0;
                hierIdx = 0;
                prelevel = hier[k].first.first->level;
            }
        }
        preNode = hier[k].first.first;
        preNode_2 = hier[k].first.second;
    }
}

__global__ void calSPScore(char* seqs, int32_t* seqInfo, int64_t* result) {
    int32_t seqNum     = seqInfo[0]; 
    int32_t seqLen     = seqInfo[1]; 
    int32_t numBlocks  = seqInfo[2]; 
    int32_t numThreads = seqInfo[3]; 
    int32_t match      = seqInfo[4]; 
    int32_t mismatch   = seqInfo[5]; 
    int32_t gap        = seqInfo[6]; 

    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int bs = blockDim.x;
    int gs = gridDim.x;
    const int threadNum = 512;
    // int tidx = bx*bs+tx;
    if (bx < numBlocks) {
        __shared__ int64_t columnScore [threadNum];
        if (tx < numThreads) {
            columnScore[tx] = 0;
        }
        __syncthreads();
        for (int l = bx; l < seqLen; l=l+gs) {
            for (int i = 0; i < seqNum-1; ++i) {
                if (tx < numThreads) {
                    for (int j = i + 1 + tx; j < seqNum; j=j+bs) {
                        if (seqs[i*seqLen+l] == 'N' || seqs[j*seqLen+l] == 'N' || 
                            seqs[i*seqLen+l] == 'n' || seqs[j*seqLen+l] == 'n' ||
                           (seqs[i*seqLen+l] == '-' && seqs[j*seqLen+l] == '-' )) {
                            columnScore[tx] += 0;
                        }
                        else if (seqs[i*seqLen+l] == '-' || seqs[j*seqLen+l] == '-') {
                            columnScore[tx] += gap;
                        }
                        else if (seqs[i*seqLen+l] == seqs[j*seqLen+l]) {
                            columnScore[tx] += match;
                        }
                        else {
                            columnScore[tx] += mismatch;
                        }
                    }
                }
                __syncthreads();
            }
        }
        for (uint32_t r = threadNum/2; r > 0; r >>= 1) {
            if (tx < r) {
                columnScore[tx]   = columnScore[tx] + columnScore[tx+r];
            }
            __syncthreads();
        }
        if (tx == 0) result[bx] = columnScore[0];
    }

}

double getSPScore_gpu(msa::utility* util, Params& param) {
    // auto scoreStart = std::chrono::high_resolution_clock::now();
    int numBlocks = 1024;
    int blockSize = 512;
    // size_t seqNum = util->memNum;
    // size_t seqLen = util->seqsLen["node_1"];
    size_t seqNum = util->seqNum;
    size_t seqLen = 0;

    while (util->seqs[0][seqLen] != 0) {
        ++seqLen;
    }
    printf("(Num, Len) - (%lu, %lu)\n", seqNum, seqLen);
    
    char*    hostSeqs = (char*)malloc(seqLen * seqNum * sizeof(char));
    int32_t* hostSeqInfo = (int32_t*)malloc(7 * sizeof(int32_t));
    int64_t* hostResult = (int64_t*)malloc(numBlocks * sizeof(int64_t));
    // int seqCount = 0;
    for (int i = 0; i < seqNum; ++i) {
        for (int j = 0; j < seqLen; ++j) {
            hostSeqs[i*seqLen+j] = util->seqs[i][j];
        }
    }
    // for (int j = 0; j < seqLen*seqNum; ++j) { 
    //     if (j%seqLen < alignment[seqCount].size()) {
    //         hostSeqs[j] = util->seqs[seqCount][j%seqLen];
    //     }
    //     else hostSeqs[j] = 0;
    //     if (j%seqLen == seqLen-1) ++seqCount;
    // }
    

    hostSeqInfo[0] = seqNum;
    hostSeqInfo[1] = seqLen;
    hostSeqInfo[2] = numBlocks;
    hostSeqInfo[3] = blockSize;
    hostSeqInfo[4] = param.match;
    hostSeqInfo[5] = param.mismatch;
    hostSeqInfo[6] = param.gapExtend;
    for (int i = 0; i < numBlocks; ++i) hostResult[i] = 0;

    char*    deviceSeqs;
    int32_t* deviceSeqInfo;
    int64_t* deviceResult;

    hipMalloc((void**)&deviceSeqs, seqLen * seqNum * sizeof(char));
    hipMalloc((void**)&deviceSeqInfo, 7 * sizeof(int32_t));
    hipMalloc((void**)&deviceResult, numBlocks * sizeof(int64_t));

    hipMemcpy(deviceSeqs, hostSeqs, seqLen * seqNum * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(deviceSeqInfo, hostSeqInfo, 7 * sizeof(int32_t), hipMemcpyHostToDevice);
    hipMemcpy(deviceResult, hostResult, numBlocks * sizeof(int64_t), hipMemcpyHostToDevice);

    calSPScore<<<numBlocks, blockSize>>>(
        deviceSeqs, 
        deviceSeqInfo,
        deviceResult
    );
    hipDeviceSynchronize();
    hipMemcpy(hostResult, deviceResult, numBlocks * sizeof(int64_t), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    int64_t gpuScore = 0;
    for (int i = 0; i < numBlocks; ++i) {
        // std::cout << "Block: "<<i<<", score: " << hostResult[i] << '\n';
        gpuScore += hostResult[i];
    }
    double score = static_cast<double>(gpuScore)/static_cast<double>(seqNum*(seqNum-1)/2);
    // std::cout << "GPU score: " << score << '\n';
    // auto scoreEnd = std::chrono::high_resolution_clock::now();
    // std::chrono::nanoseconds scoreTime = scoreEnd - scoreStart;
    
    // printf("GPU score: %f, Runtime %d ms\n", score, scoreTime.count() / 1000000);
    
    hipFree(deviceSeqs);
    hipFree(deviceSeqInfo);
    hipFree(deviceResult);
    free(hostSeqs);
    free(hostSeqInfo);
    free(hostResult);
    
    return score;
} 

double getSPScore_cpu(std::vector<std::string>& alignment, Params& param) {
    auto scoreStart = std::chrono::high_resolution_clock::now();
    size_t seqNum = alignment.size();
    size_t seqLen = alignment[0].size();
    printf("(Num, Len) - (%lu, %lu)\n", seqNum, seqLen);
    double score = 0;
    for (int l = 0; l < seqLen; l++) {
        double columnScore = 0;
        for (int i = 0; i < seqNum - 1; ++i) {
            for (int j = i + 1; j < seqNum; ++j) {
                if (alignment[i][l] == 'N' || alignment[j][l] == 'N' ||
                   (alignment[i][l] == '-' && alignment[j][l] == '-' )) {
                    columnScore += 0;
                }
                else if (alignment[i][l] == '-' || alignment[j][l] == '-') {
                    columnScore += static_cast<double>(param.gapExtend);
                }
                else if (alignment[i][l] == alignment[j][l]) {
                    columnScore +=  static_cast<double>(param.match);
                }
                else {
                    columnScore +=  static_cast<double>(param.mismatch);
                }
            }
        }
        int totalPairs = seqNum*(seqNum-1)/2;
        columnScore /= static_cast<double>(totalPairs);
        score += columnScore;
        if (l % 100 == 0) {
            auto scoreEnd = std::chrono::high_resolution_clock::now();
            std::chrono::nanoseconds scoreTime = scoreEnd - scoreStart;
            printf("CPU score: %f, Runtime %ld ms\n", score, scoreTime.count() / 1000000);
        }
    }
    auto scoreEnd = std::chrono::high_resolution_clock::now();
    std::chrono::nanoseconds scoreTime = scoreEnd - scoreStart;
    printf("CPU score: %f, Runtime %ld ms\n", score, scoreTime.count() / 1000000);
    return score;
} 



// Regressive Method (not maintained)

/*
void getLongestDescendent(Tree* tree, msa::utility* util) {
    std::stack<Node*> postOrder;
    getPostOrderList(tree->root, postOrder);
    while (!postOrder.empty()) {
        Node* n = postOrder.top();
        postOrder.pop();
        if (n->children.empty()) n->setLongestDescendant(n);
        else if (n->children.size() == 1) n->setLongestDescendant(n->children[0]->longestDescendant);
        else {
            Node* longest = n->children[0]->longestDescendant;
            for (int i = 1; i < n->children.size(); ++i) {
                if (util->seqs[n->children[i]->longestDescendant->identifier].size() > util->seqs[longest->identifier].size()) {
                    longest = n->children[i]->longestDescendant;
                }
            }
            n->setLongestDescendant(longest);
        }
    }
    return;

}

void getSubMSANodes(std::vector<Node*>& subMSANodes, Node* startNode, int N) {
    if (startNode->is_leaf()) return;
    std::queue<Node*> visited;
    std::queue<Node*> added;
    visited.push(startNode);
    int minNumNodes = N;
    while (visited.size() < minNumNodes && !visited.empty()) {
        
        Node* currentNode = visited.front();
        visited.pop();
        // std::cout << "start: " << currentNode->identifier << '\n';
        if (!currentNode->children.empty()) {
            for (auto ch: currentNode->children) {
                // std::cout << "ch: " << ch->identifier << '\n';
                visited.push(ch);
            }
        }
        else {
            added.push(currentNode);
            minNumNodes -= 1;
        }
    }
    while (!visited.empty()) {
        added.push(visited.front());
        visited.pop();
    }
    while (!added.empty()) {
        subMSANodes.push_back(added.front());
        added.pop();
    }
    // for (auto a: subMSANodes) std::cout << a->longestDescendant->identifier << ',';
    // std::cout << '\n';
    return;
    
}

void getSubMSAs(std::map<int, std::vector<std::vector<Node*>>>& subMSAs, Tree* T, int N) {
    // subMSAs (Level, clusters)
    subMSAs.clear();
    int level = 0;
    Node* subRoot = T->root;
    std::vector<Node*> parentMSA;
    std::vector<std::pair<int, std::vector<Node*>>> subMSAList;
    getSubMSANodes(parentMSA, subRoot, N);
    subMSAList.push_back(std::make_pair(level, parentMSA));
    int start = 0, end = subMSAList.size();
    level++;
    while (true) {
        for (int idx = start; idx < end; ++idx) {
            for (auto node: subMSAList[idx].second) {
                // std::cout << node->identifier << '\n';
                std::vector<Node*> childMSA;
                getSubMSANodes(childMSA, node, N);
                if (!childMSA.empty()) {
                    subMSAList.push_back(std::make_pair(level, childMSA));
                }
            }
        }
        start = end;
        end = subMSAList.size();
        if (start == end) break;
        ++level;
    }
    for (int i = 0; i < level; ++i) {
        std::vector<std::vector<Node*>> temp;
        subMSAs[i] = temp;
    }
    for (auto msa: subMSAList) {
        subMSAs[msa.first].push_back(msa.second);
    }
    // for (auto it = subMSAs.begin(); it != subMSAs.end(); ++it) {
    //     std::cout << "Level: " << it->first << '\n';
    //     for (auto s: it->second) {
    //         for (auto ss: s) std::cout << ss->longestDescendant->identifier << ',';
    //         std::cout << '\n';
    //     }
    // }
    return;
}

void getAlnPairs(std::vector<std::vector<std::pair<Node*, Node*>>>& alnPairs, std::vector<std::vector<Node*>>& clusters) {
    std::vector<std::vector<Node*>> currentLevel, nextLevel;
    currentLevel = clusters;
    while (!currentLevel.empty()) {
        std::vector<std::pair<Node*, Node*>> temp;
        int cidx = 0;
        for (auto cluster: currentLevel) {
            // std::cout << cidx << ':' << cluster.size() <<'\n';
            if (cluster.size() == 1) continue;
            std::vector<Node*> nextLevelTemp;
            for (int i = 0; i < cluster.size()-1; i+=2) {
                nextLevelTemp.push_back(cluster[i]);
                temp.push_back(std::make_pair(cluster[i], cluster[i+1]));
            }
            if (cluster.size()%2 == 1) nextLevelTemp.push_back(cluster.back());
            nextLevel.push_back(nextLevelTemp);
        }
        alnPairs.push_back(temp);
        // std::cout << nextLevel.size() << '\n';
        currentLevel = nextLevel;
        nextLevel.clear();
    }
    return;
}

void storeMSA(Tree* T, std::vector<Node*>& nodes, msa::utility* util, int level) {
    for (auto n: nodes) {
        assert(n->msa.size() == n->msaIdx.size());
        for (int i = 0; i < n->msa.size(); ++i) {
            int sIdx = n->msaIdx[i];
            std::string sIdentifier = n->msa[i];
            int storage = util->seqsStorage[sIdx];
            int j = 0;
            std::string msa = "";
            while (util->seqBuf[storage][sIdx][j] != 0) {
                msa += util->seqBuf[storage][sIdx][j];
                ++j;
            }
            T->allNodes[sIdentifier]->msaSeq[level] = msa;
        }
    }
    return;
}

void resetSeqMem(std::vector<Node*>& nodes, msa::utility* util) {
    for (auto n: nodes) {
        assert(n->msa.size() == n->msaIdx.size());
        for (int i = 0; i < n->msa.size(); ++i) {
            int sIdx = n->msaIdx[i];
            std::string sIdentifier = n->msa[i];
            int storage = util->seqsStorage[sIdx];
            std::string rawSeq = util->seqs[sIdentifier];
            for (int j = 0; j < util->memLen; ++j) {
                if (j < rawSeq.size()) util->seqBuf[storage][sIdx][j] = rawSeq[j];
                else util->seqBuf[storage][sIdx][j] = 0;
            }
            storage = 1 - storage;
            for (int j = 0; j < util->memLen; ++j) {
                util->seqBuf[storage][sIdx][j] = 0;
            }
        }
    }
    return;
}

std::string removeGap(std::string s) {
    std::string s_noGap = "";
    for (int i = 0; i < s.size(); ++i) if (s[i] != '-') s_noGap += s[i];
    return s_noGap;
}

void merger_ref(Tree* tree, std::map<int, Node*>& refNodes, msa::utility* util, std::string& refString, std::string& qryString, int qLevel) {
    int rIdx = 0, qIdx = 0, alnIdx = 0;
    int refLen = refString.size(), qryLen = qryString.size();
    if (removeGap(refString) != removeGap(qryString)) {
        std::cout << "Unmatched Seq.\n"; exit(1);
    }
    
    while (rIdx < refLen && qIdx < qryLen) {
        if (alnIdx > refLen && alnIdx > qryLen) util->memCheck(alnIdx);
        if (refString[rIdx] == qryString[qIdx] && refString[rIdx] != '-') {
            for (auto n: refNodes) {
                int sIdx = n.first;
                int storeFrom = util->seqsStorage[sIdx];
                int storeTo = 1 - storeFrom;
                util->seqBuf[storeTo][sIdx][alnIdx] = util->seqBuf[storeFrom][sIdx][rIdx];
            }
            ++alnIdx;++qIdx;++rIdx;
        }
        else if (refString[rIdx] == '-' && qryString[qIdx] != '-') {
            int consecGap = 0;
            int k = rIdx;
            while (refString[k] == '-' && k < refLen) {
                ++consecGap;
                ++k;
            }
            for (size_t g = 0; g < consecGap; ++g) {
                for (auto n: refNodes) {
                    int sIdx = n.first;
                    int storeFrom = util->seqsStorage[sIdx];
                    int storeTo = 1 - storeFrom;
                    util->seqBuf[storeTo][sIdx][alnIdx] = util->seqBuf[storeFrom][sIdx][rIdx];
                }
                ++alnIdx;++rIdx;
            }
        }
        else if (refString[rIdx] != '-' && qryString[qIdx] == '-') {
            int consecGap = 0;
            int k = qIdx;
            while (qryString[k] == '-' && k < qryLen) {
                ++consecGap;
                ++k;
            }
            for (size_t g = 0; g < consecGap; ++g) {
                for (auto n: refNodes) {
                    int sIdx = n.first;
                    int storeFrom = util->seqsStorage[sIdx];
                    int storeTo = 1 - storeFrom;
                    util->seqBuf[storeTo][sIdx][alnIdx] = '-';
                }
                ++alnIdx;++qIdx;
            }
        }
        else {
            int consecGap = 0;
            int kr = rIdx, kq = qIdx;
            while (refString[kr] == '-' && kr < refLen) {
                ++consecGap;
                ++kr;
            }
            for (size_t g = 0; g < consecGap; ++g) {
                for (auto n: refNodes) {
                    int sIdx = n.first;
                    int storeFrom = util->seqsStorage[sIdx];
                    int storeTo = 1 - storeFrom;
                    util->seqBuf[storeTo][sIdx][alnIdx] = util->seqBuf[storeFrom][sIdx][rIdx];
                }
                ++alnIdx;++rIdx;
            }
            consecGap = 0;
            while (qryString[kq] == '-' && kq < qryLen) {
                ++consecGap;
                ++kq;
            }
            for (size_t g = 0; g < consecGap; ++g) {
                for (auto n: refNodes) {
                    int sIdx = n.first;
                    int storeFrom = util->seqsStorage[sIdx];
                    int storeTo = 1 - storeFrom;
                    util->seqBuf[storeTo][sIdx][alnIdx] = '-';
                }
                ++alnIdx;++qIdx;
            }
        }
    }
    while (rIdx < refLen) {
        for (auto n: refNodes) {
            int sIdx = n.first;
            int storeFrom = util->seqsStorage[sIdx];
            int storeTo = 1 - storeFrom;
            util->seqBuf[storeTo][sIdx][alnIdx] = util->seqBuf[storeFrom][sIdx][rIdx];
        }
        ++alnIdx;++rIdx;
    }
    while (qIdx < qryLen) {
        for (auto n: refNodes) {
            int sIdx = n.first;
            int storeFrom = util->seqsStorage[sIdx];
            int storeTo = 1 - storeFrom;
            util->seqBuf[storeTo][sIdx][alnIdx] = '-';
        }
        ++alnIdx;++qIdx;
    }
    for (auto n: refNodes) {
        int sIdx = n.first;
        util->changeStorage(sIdx);
    }
    return;
}

void merger_qry(Tree* tree, std::vector<Node*>& qryNodes, msa::utility* util, std::string& refString, std::string& qryString, int qLevel) {
    int rIdx = 0, qIdx = 0, alnIdx = 0;
    int refLen = refString.size(), qryLen = qryString.size();
    // std::cout << refString << '\n' << qryString << '\n';
    if (removeGap(refString) != removeGap(qryString)) {
        std::cout << "XXXXXX\n";
    }
    std::map<int, int> qrySeqIdx;
    for (auto n: qryNodes) qrySeqIdx[util->seqsIdx[n->longestDescendant->identifier]] = util->seqsStorage[util->seqsIdx[n->longestDescendant->identifier]];
    // for (auto n: qrySeqIdx) std::cout << n.first << ',';
    // std::cout << '\n';
    assert (refLen >= qryLen);
    for (int i = 0; i < refLen; ++i) {
        if (refString[i] == qryString[qIdx]) {
            for (auto n: qryNodes) {
                int sIdx = util->seqsIdx[n->longestDescendant->identifier];
                int storeFrom = util->seqsStorage[sIdx];
                int storeTo = 1 - storeFrom;
                // util->seqBuf[storeTo][sIdx][i] = util->seqBuf[storeFrom][sIdx][qIdx];
                util->seqBuf[storeTo][sIdx][i] = n->longestDescendant->msaSeq[qLevel][qIdx];
            }
            ++qIdx;
        }
        else {
            for (auto n: qryNodes) {
                int sIdx = util->seqsIdx[n->longestDescendant->identifier];
                int storeFrom = util->seqsStorage[sIdx];
                int storeTo = 1 - storeFrom;
                util->seqBuf[storeTo][sIdx][i] = '-';
            }
        }
    }   
    for (auto n: qryNodes) {
        int sIdx = util->seqsIdx[n->longestDescendant->identifier];
        util->changeStorage(sIdx);
    }
    return;
}

void transitivityMerge_regressive(Tree* tree, std::map<int, std::vector<std::vector<Node*>>>& subMSAs, msa::utility* util) {

    auto mergeStart = std::chrono::high_resolution_clock::now();
    // clear seq storage
    for (int sIdx = 0; sIdx < util->memNum; ++sIdx) {
        util->seqsStorage[sIdx] = 0;
        for (int i = 0; i < util->memLen; ++i) {
            util->seqBuf[0][sIdx][i] = 0;
            util->seqBuf[1][sIdx][i] = 0;
        }
    }
    // store MSA to storage
    assert(subMSAs[0].size() == 1);
    for (auto n: tree->allNodes) {
        if (n.second->is_leaf()) {
            int sIdx = util->seqsIdx[n.second->longestDescendant->identifier];
            std::string seq = tree->allNodes[n.second->longestDescendant->identifier]->msaSeq.begin()->second;
            for (int i = 0; i < seq.size(); ++i) {
                util->seqBuf[0][sIdx][i] = seq[i];
            }
        }
    }
    
    // for (int i = 0; i < util->memNum; ++i) {
    //     int storage = util->seqsStorage[i];
    //     int s = 0;
    //     while (true) {
    //         if (util->seqBuf[storage][i][s] == 0) break;
    //         ++s;
    //     }
    //     std::cout << i << ':' << s << '\n';
    // }
    std::map<int, Node*> refNodes;
    
    for (auto n: subMSAs[0][0]) {
        int longestIdx = util->seqsIdx[n->longestDescendant->identifier];
        if (refNodes.find(longestIdx) == refNodes.end()) {
            refNodes[longestIdx] = n->longestDescendant;
        }
    }
    for (int i = 1; i < subMSAs.size(); ++i) {
        auto parentMSAs = subMSAs[i-1];
        int childIdx = 0;
        int parentIdx = 0;
        
        for (auto pMSA: parentMSAs) {
            for (auto node: pMSA) {
                auto longestD = node->longestDescendant;
                while (childIdx < subMSAs[i].size()) {
                    auto cMSA = subMSAs[i][childIdx];
                    bool is_child = false;
                    for (auto cNode: cMSA) {
                        if (cNode->longestDescendant->identifier == longestD->identifier) {
                            is_child = true;
                            break;
                        }
                    }
                    if (is_child) {
                        printf("Merge Ref: %d-%d (qry) to %d-%d (ref).\n", i, childIdx, i-1, parentIdx); 
                        std::string refString = "";
                        int sIdx = util->seqsIdx[longestD->identifier];
                        int storage = util->seqsStorage[sIdx];
                        int s = 0;
                        while (true) {
                            if (util->seqBuf[storage][sIdx][s] == 0) break;
                            refString += util->seqBuf[storage][sIdx][s];
                            ++s;
                        }
                        std::string qryString = tree->allNodes[longestD->identifier]->msaSeq[i];
                        merger_ref(tree, refNodes, util, refString, qryString, i);
                        ++childIdx;
                        
                        // debug
                        // std::string refString_post = "";
                        // sIdx = util->seqsIdx[longestD->identifier];
                        // storage = util->seqsStorage[sIdx];
                        // s = 0;
                        // while (true) {
                        //     if (util->seqBuf[storage][sIdx][s] == 0) break;
                        //     refString_post += util->seqBuf[storage][sIdx][s];
                        //     ++s;
                        // }
                        // std::string rawr = "", rawq = "";
                        // for (int k = 0; k < refString.size(); ++k) if (refString[k] != '-') rawr += refString[k];
                        // for (int k = 0; k < refString_post.size(); ++k) if (refString_post[k] != '-') rawq += refString_post[k];
                        // if (rawr != rawq) {
                            
                        //     std::cout << "Post: Unmatched Seq.\n";
                        //     std::cout << refString << '\n' << qryString << '\n';
                        //     std::cout << refString_post << '\n';
                        //     std::cout << rawr << '\n' << rawq << '\n';
                        //     exit(1);
                        // }
                    }
                    else break;
                }
            }
            ++parentIdx;
        }

        childIdx = 0; parentIdx = 0;
        
        for (auto pMSA: parentMSAs) {
            for (auto node: pMSA) {
                auto longestD = node->longestDescendant;
                while (childIdx < subMSAs[i].size()) {
                    auto cMSA = subMSAs[i][childIdx];
                    bool is_child = false;
                    for (auto cNode: cMSA) {
                        if (cNode->longestDescendant->identifier == longestD->identifier) {
                            is_child = true;
                            break;
                        }
                    }
                    if (is_child) {
                        printf("Merge Qry: %d-%d (qry) to %d-%d (ref).\n", i, childIdx, i-1, parentIdx); 
                        std::string refString = "";
                        int sIdx = util->seqsIdx[longestD->identifier];
                        int storage = util->seqsStorage[sIdx];
                        int s = 0;
                        while (true) {
                            if (util->seqBuf[storage][sIdx][s] == 0) break;
                            refString += util->seqBuf[storage][sIdx][s];
                            ++s;
                        }
                        std::string qryString = tree->allNodes[longestD->identifier]->msaSeq[i];
                        merger_qry(tree, cMSA, util, refString, qryString, i);
                        ++childIdx;
                        // debug
                        // std::string refString_post = "";
                        // sIdx = util->seqsIdx[longestD->identifier];
                        // storage = util->seqsStorage[sIdx];
                        // s = 0;
                        // while (true) {
                        //     if (util->seqBuf[storage][sIdx][s] == 0) break;
                        //     refString_post += util->seqBuf[storage][sIdx][s];
                        //     ++s;
                        // }
                        // std::string rawr = "", rawq = "";
                        // for (int k = 0; k < refString.size(); ++k) if (refString[k] != '-') rawr += refString[k];
                        // for (int k = 0; k < refString_post.size(); ++k) if (refString_post[k] != '-') rawq += refString_post[k];
                        // if (rawr != rawq) {
                            
                        //     std::cout << "PostMergeQ: Unmatched Seq.\n";
                        //     std::cout << refString << '\n' << qryString << '\n';
                        //     std::cout << refString_post << '\n';
                        //     exit(1);
                        // } 
                        for (auto n: cMSA) {
                            int longestIdx = util->seqsIdx[n->longestDescendant->identifier];
                            if (refNodes.find(longestIdx) == refNodes.end()) {
                                refNodes[longestIdx] = n->longestDescendant;
                            }
                        }
                    }
                    else break;
                }
            }
            ++parentIdx;
        }
    }
    auto mergeEnd = std::chrono::high_resolution_clock::now();
    std::chrono::nanoseconds mergeTime = mergeEnd - mergeStart;
    printf("Merge time : %d us\n",  (mergeTime.count() / 1000));      
    return;
}

void msaPostOrderTraversal_multigpu_regressive(Tree* tree, std::vector<std::pair<Node*, Node*>> nodes, msa::utility* util, Params& param)
{
    auto freqStart = std::chrono::high_resolution_clock::now();
    for (auto n_pair: nodes) {
        auto n = std::make_pair(tree->allNodes[n_pair.first->identifier], tree->allNodes[n_pair.second->identifier]);
        if (n.first->msaIdx.size() == 0) {
            n.first->msaIdx.push_back(util->seqsIdx[n.first->longestDescendant->identifier]);
            n.first->msa.push_back(n.first->longestDescendant->identifier);
            util->seqsLen[n.first->identifier] = util->seqsLen[n.first->longestDescendant->identifier];
        }
        if (n.second->msaIdx.size() == 0) {
            n.second->msaIdx.push_back(util->seqsIdx[n.second->longestDescendant->identifier]);
            n.second->msa.push_back(n.second->longestDescendant->identifier);
            util->seqsLen[n.second->identifier] = util->seqsLen[n.second->longestDescendant->identifier];
        }
        // std::cout << n.first->identifier << ':' << n.second->identifier << '\n';
        // for (auto id: n.first->msaIdx) std::cout << id << ',';
        // std::cout << '\n';
        // for (auto id: n.second->msaIdx) std::cout << id << ',';
        // std::cout << '\n';
    }

    int numBlocks = 1024; 
    int blockSize = THREAD_NUM;
    int gpuNum;
    hipGetDeviceCount(&gpuNum); // number of CUDA devices
    
    // get maximum sequence/profile length 
    int32_t seqLen = 0;
    for (auto n: nodes) {
        int32_t refLen = util->seqsLen[n.first->identifier];
        int32_t qryLen = util->seqsLen[n.second->identifier];
        int32_t tempMax = max(qryLen, refLen);
        seqLen = max(seqLen, tempMax);
    }
    
    int roundGPU = nodes.size() / numBlocks + 1;
    if (nodes.size()%numBlocks == 0) roundGPU -= 1;
    if (roundGPU < gpuNum) gpuNum = roundGPU;

    int* alignSize = new int[roundGPU];
    int32_t* seqNum = new int32_t[roundGPU];
    uint16_t** hostFreq = new uint16_t* [roundGPU];
    int8_t**   hostAln = new int8_t* [roundGPU];
    int32_t**  hostLen = new int32_t* [roundGPU];
    int32_t**  hostAlnLen = new int32_t* [roundGPU];
    int32_t**  hostSeqInfo = new int32_t* [roundGPU];
    
    paramType* hostParam = (paramType*)malloc(28 * sizeof(paramType)); 

    if (param.scoreMode == 0) {
        for (int i = 0; i < 5; ++i) {
            for (int j = 0; j < 5; ++j) {
                if (i == 5 || j == 5)          hostParam[i*5+j] = 0;
                else if (i == j)               hostParam[i*5+j] = param.match;
                else if (i-j == 2 || j-i == 2) hostParam[i*5+j] = param.trans;
                else                           hostParam[i*5+j] = param.mismatch;
            }
        }
        hostParam[25] = param.gapOpen;
        hostParam[26] = param.gapExtend;
        hostParam[27] = param.xdrop;
    }
    else if (param.scoreMode == 1) {
        for (int i = 0; i < 5; ++i) for (int j = 0; j < 5; ++j) hostParam[i*5+j] = param.hoxd70[i][j];
        hostParam[25] = param.hoxd70_gapOpen;
        hostParam[26] = param.hoxd70_gapExtend;
        hostParam[27] = param.xdrop;
    }
    
    
    std::vector<std::vector<uint16_t*>> freq;
    std::vector<std::vector<std::pair<int32_t, int32_t>>> seqIdx;
    std::vector<std::vector<std::pair<int32_t, int32_t>>> len;
    for (int rn = 0; rn < roundGPU; ++rn) {
        int pairsLeft = nodes.size() - rn*numBlocks;
        if (pairsLeft < numBlocks) alignSize[rn] = pairsLeft;
        else alignSize[rn] = numBlocks;
        seqNum[rn] = 0;
        hostFreq[rn] = (uint16_t*)malloc(12*seqLen * alignSize[rn] * sizeof(uint16_t));
        hostAln[rn] = (int8_t*)malloc(2*seqLen * alignSize[rn] * sizeof(int8_t));
        hostLen[rn] = (int32_t*)malloc(2*alignSize[rn] * sizeof(int32_t));
        hostAlnLen[rn] = (int32_t*)malloc(alignSize[rn] * sizeof(int32_t));
        hostSeqInfo[rn] = (int32_t*)malloc(5 * sizeof(int32_t));
        // store all sequences to array
        std::vector<uint16_t*> freqTemp;
        std::vector<std::pair<int32_t, int32_t>> seqIdxTemp;
        std::vector<std::pair<int32_t, int32_t>> lenTemp;
        for (int n = 0; n < alignSize[rn]; ++n) {
            int32_t nIdx = n + rn*numBlocks;
            int32_t qryIdx = 0;
            int32_t refIdx = 0;
            int32_t refLen = util->seqsLen[nodes[nIdx].first->identifier];
            int32_t qryLen = util->seqsLen[nodes[nIdx].second->identifier];
            refIdx = seqNum[rn];
            uint16_t *temp = new uint16_t[12*seqLen]; 
            for (int i = 0; i < 12*seqLen; ++i) temp[i]=0;
            // assert(temp.size() == 12*seqLen);
            // tbb::blocked_range<int> rangeRef(0, refLen);
            for (auto sIdx: tree->allNodes[nodes[nIdx].first->identifier]->msaIdx) { 
                int storage = util->seqsStorage[sIdx];
                tbb::parallel_for(tbb::blocked_range<int>(0, refLen), [&](tbb::blocked_range<int> r) {
                for (int s = r.begin(); s < r.end(); ++s) {
                    if      (util->seqBuf[storage][sIdx][s] == 'A' || util->seqBuf[storage][sIdx][s] == 'a') temp[6*s+0]+=1;
                    else if (util->seqBuf[storage][sIdx][s] == 'C' || util->seqBuf[storage][sIdx][s] == 'c') temp[6*s+1]+=1;
                    else if (util->seqBuf[storage][sIdx][s] == 'G' || util->seqBuf[storage][sIdx][s] == 'g') temp[6*s+2]+=1;
                    else if (util->seqBuf[storage][sIdx][s] == 'T' || util->seqBuf[storage][sIdx][s] == 't' ||
                             util->seqBuf[storage][sIdx][s] == 'U' || util->seqBuf[storage][sIdx][s] == 'u') temp[6*s+3]+=1;
                    else if (util->seqBuf[storage][sIdx][s] == 'N' || util->seqBuf[storage][sIdx][s] == 'n') temp[6*s+4]+=1;
                    else                                                                                     temp[6*s+5]+=1;
                }
                });
                seqNum[rn] += 1;
            }
            qryIdx = seqNum[rn];
            for (auto sIdx: tree->allNodes[nodes[nIdx].second->identifier]->msaIdx) { 
                int storage = util->seqsStorage[sIdx];
                tbb::parallel_for(tbb::blocked_range<int>(0, qryLen), [&](tbb::blocked_range<int> r) {
                for (int s = r.begin(); s < r.end(); ++s) {
                    if      (util->seqBuf[storage][sIdx][s] == 'A' || util->seqBuf[storage][sIdx][s] == 'a') temp[6*(seqLen+s)+0]+=1;
                    else if (util->seqBuf[storage][sIdx][s] == 'C' || util->seqBuf[storage][sIdx][s] == 'c') temp[6*(seqLen+s)+1]+=1;
                    else if (util->seqBuf[storage][sIdx][s] == 'G' || util->seqBuf[storage][sIdx][s] == 'g') temp[6*(seqLen+s)+2]+=1;
                    else if (util->seqBuf[storage][sIdx][s] == 'T' || util->seqBuf[storage][sIdx][s] == 't' ||
                             util->seqBuf[storage][sIdx][s] == 'U' || util->seqBuf[storage][sIdx][s] == 'u') temp[6*(seqLen+s)+3]+=1;
                    else if (util->seqBuf[storage][sIdx][s] == 'N' || util->seqBuf[storage][sIdx][s] == 'n') temp[6*(seqLen+s)+4]+=1;
                    else                                                                     temp[6*(seqLen+s)+5]+=1;
                }
                });
                seqNum[rn] += 1;
            }
            seqIdxTemp.push_back(std::make_pair(refIdx, qryIdx));
            lenTemp.push_back(std::make_pair(refLen, qryLen));
            freqTemp.push_back(temp);
        }
        freq.push_back(freqTemp);
        len.push_back(lenTemp);
        seqIdx.push_back(seqIdxTemp);
    }

    tbb::parallel_for(tbb::blocked_range<int>(0, roundGPU), [&](tbb::blocked_range<int> range){
        for (int gn = range.begin(); gn < range.end(); ++gn) {
            for (int j = 0; j < 2*alignSize[gn]; ++j) { 
                if (j%2 == 0) hostLen[gn][j] = len[gn][j/2].first;
                else          hostLen[gn][j] = len[gn][j/2].second;
            }
            for (int j = 0; j < alignSize[gn]; ++j) {
                for (int l = 0; l < 12*seqLen; ++l) {
                    hostFreq[gn][12*seqLen*j+l] = freq[gn][j][l];
                }
            }
            for (int j = 0; j < 2*seqLen*alignSize[gn]; ++j) { 
                hostAln[gn][j] = 0;
            }
            for (int j = 0; j < alignSize[gn]; ++j) { 
                hostAlnLen[gn][j] = 0;
            }
            hostSeqInfo[gn][0] = seqLen;
            hostSeqInfo[gn][1] = seqNum[gn];
            hostSeqInfo[gn][2] = alignSize[gn];
            hostSeqInfo[gn][3] = numBlocks;
            hostSeqInfo[gn][4] = param.scoreMode;
        }
    });

    auto freqEnd = std::chrono::high_resolution_clock::now();
    std::chrono::nanoseconds freqTime = freqEnd -freqStart;
    printf("Preprocessing time : %d ms\n",  (freqTime.count() / 1000000));        

    auto kernelStart = std::chrono::high_resolution_clock::now();
    uint16_t** deviceFreq = new uint16_t* [gpuNum];
    int8_t**   deviceAln = new int8_t* [gpuNum];
    int32_t**  deviceLen = new int32_t* [gpuNum];
    int32_t**  deviceAlnLen = new int32_t* [gpuNum];
    int32_t**  deviceSeqInfo = new int32_t* [gpuNum];
    paramType**  deviceParam = new paramType* [gpuNum];
    // int nowRound = 0;
    std::atomic<int> nowRound;
    nowRound.store(0);
    tbb::parallel_for(tbb::blocked_range<int>(0, gpuNum), [&](tbb::blocked_range<int> range){ 
        for (int gn = range.begin(); gn < range.end(); ++gn) {
            hipSetDevice(gn);
            int nowMemSize = alignSize[gn];
            // hipError_t error;
            hipMalloc((void**)&deviceFreq[gn], 12*seqLen * alignSize[gn] * sizeof(uint16_t));
            // error = hipGetLastError(); printf("CUDA error Freq1: %s, %d\n, E",hipGetErrorString(error), gn); 
            hipMalloc((void**)&deviceAln[gn], 2*seqLen * alignSize[gn] * sizeof(int8_t));
            // error = hipGetLastError(); printf("CUDA error Freq2: %s\n",hipGetErrorString(error)); 
            hipMalloc((void**)&deviceLen[gn], 2*alignSize[gn] * sizeof(int32_t));
            // error = hipGetLastError(); printf("CUDA error Freq3: %s\n",hipGetErrorString(error)); 
            hipMalloc((void**)&deviceAlnLen[gn], alignSize[gn] * sizeof(int32_t));
            // error = hipGetLastError(); printf("CUDA error Freq4: %s\n",hipGetErrorString(error)); 
            hipMalloc((void**)&deviceSeqInfo[gn], 5 * sizeof(int32_t));
            // error = hipGetLastError(); printf("CUDA error Freq5: %s\n",hipGetErrorString(error)); 
            hipMalloc((void**)&deviceParam[gn], 28 * sizeof(paramType));
            hipMemcpy(deviceParam[gn], hostParam, 28 * sizeof(paramType), hipMemcpyHostToDevice);
            // error = hipGetLastError(); printf("CUDA error Freq6: %s\n",hipGetErrorString(error)); 
                
            while (nowRound < roundGPU) {
                int rn = nowRound.fetch_add(1);
                if (alignSize[rn] != nowMemSize) {
                    // hipSetDevice(gn);
                    hipFree(deviceFreq[gn]);
                    hipFree(deviceAln[gn]);
                    hipFree(deviceLen[gn]);
                    hipFree(deviceAlnLen[gn]);
                    // hipFree(deviceParam[gn]);
                    // hipFree(deviceSeqInfo[gn]);
                    // error = hipGetLastError(); printf("CUDA error Free: %s\n",hipGetErrorString(error)); 
                    hipDeviceSynchronize();
                    hipMalloc((void**)&deviceFreq[gn], 12*seqLen*alignSize[rn] * sizeof(uint16_t));
                    hipMalloc((void**)&deviceAln[gn], 2*seqLen*alignSize[rn] * sizeof(int8_t));
                    hipMalloc((void**)&deviceLen[gn], 2*alignSize[rn] * sizeof(int32_t));
                    hipMalloc((void**)&deviceAlnLen[gn], alignSize[rn] * sizeof(int32_t));
                    // hipMalloc((void**)&deviceSeqInfo[gn], 5 * sizeof(int32_t));
                    // error = hipGetLastError(); printf("CUDA error Alloc: %s\n",hipGetErrorString(error)); 
                }
                
                hipMemcpy(deviceFreq[gn], hostFreq[rn], 12*seqLen * alignSize[rn] * sizeof(uint16_t), hipMemcpyHostToDevice);
                // error = hipGetLastError(); printf("CUDA error Freq: %s\n",hipGetErrorString(error)); 
                hipMemcpy(deviceAln[gn], hostAln[rn], 2*seqLen * alignSize[rn] * sizeof(int8_t), hipMemcpyHostToDevice);
                // error = hipGetLastError(); printf("CUDA error Aln: %s\n",hipGetErrorString(error)); 
                hipMemcpy(deviceLen[gn], hostLen[rn], 2*alignSize[rn] * sizeof(int32_t), hipMemcpyHostToDevice);
                // error = hipGetLastError(); printf("CUDA error Len: %s\n",hipGetErrorString(error)); 
                hipMemcpy(deviceAlnLen[gn], hostAlnLen[rn], alignSize[rn] * sizeof(int32_t), hipMemcpyHostToDevice);
                // error = hipGetLastError(); printf("CUDA error AlnLen: %s\n",hipGetErrorString(error)); 
                hipMemcpy(deviceSeqInfo[gn], hostSeqInfo[rn], 5 * sizeof(int32_t), hipMemcpyHostToDevice);
                // error = hipGetLastError(); printf("CUDA error SeqInfo: %s\n",hipGetErrorString(error)); 
                // hipMemcpy(deviceParam[gn], hostParam[rn], 7 * sizeof(paramType), hipMemcpyHostToDevice);
                // error = hipGetLastError(); printf("CUDA error Param: %s\n",hipGetErrorString(error)); 
                std::string berr = hipGetErrorString(hipGetLastError());
                if (berr != "no error") printf("ERROR: Before kernel %s!\n", berr.c_str());
                alignGrpToGrp_talco<<<numBlocks, blockSize>>>(
                    deviceFreq[gn],
                    deviceAln[gn], 
                    deviceLen[gn],
                    deviceAlnLen[gn],
                    deviceSeqInfo[gn], 
                    deviceParam[gn]
                );
                hipDeviceSynchronize();
                std::string aerr = hipGetErrorString(hipGetLastError());
                if (aerr != "no error") printf("ERROR: After kernel %s!\n", aerr.c_str());
                hipMemcpy(hostAln[rn], deviceAln[gn], 2*seqLen * alignSize[rn] * sizeof(int8_t), hipMemcpyDeviceToHost);
                // error = hipGetLastError(); printf("CUDA error rAln: %s\n",hipGetErrorString(error)); 
                hipMemcpy(hostAlnLen[rn], deviceAlnLen[gn], alignSize[rn] * sizeof(int32_t), hipMemcpyDeviceToHost);
                hipDeviceSynchronize();  
            }
        }
    });

    // free memory  
    for (int gn = 0; gn < gpuNum; ++gn) {
        hipSetDevice(gn);
        hipFree(deviceFreq[gn]);
        hipFree(deviceAlnLen[gn]);
        hipFree(deviceLen[gn]);
        hipFree(deviceAln[gn]);
        hipFree(deviceParam[gn]);
        hipFree(deviceSeqInfo[gn]);
        hipDeviceSynchronize();  
    }

    auto kernelEnd = std::chrono::high_resolution_clock::now();
    std::chrono::nanoseconds kernelTime = kernelEnd - kernelStart;
    int totalPairs = 0;
    for (int gn = 0; gn < roundGPU; ++gn) totalPairs += alignSize[gn];
    std::cout << "GPU KernelTime "<< kernelTime.count() / 1000000<< " ms\n";

    auto reAlnStart = std::chrono::high_resolution_clock::now();
    
    int maxAlnLen = 0;
    for (int gn = 0; gn < roundGPU; ++gn) {
       for (int k = 0; k <  alignSize[gn]; ++k) {
            if (hostAlnLen[gn][k] > maxAlnLen) maxAlnLen = hostAlnLen[gn][k];
        }
    }
    util->memCheck(maxAlnLen);
        
    for (int gn = 0; gn < roundGPU; ++gn) {
        if (alignSize[gn] == 0) break;
        tbb::parallel_for(tbb::blocked_range<int>(0, alignSize[gn]), [&](tbb::blocked_range<int> range) {
            // for (int k = 0; k < alignSize[gn]; ++k) {
            for (int k = range.begin(); k < range.end(); ++k) {
                // std::vector<std::string> alignment;
                int32_t refNum = seqIdx[gn][k].second - seqIdx[gn][k].first;
                int32_t qryNum = (k !=  alignSize[gn]-1) ? seqIdx[gn][k+1].first - seqIdx[gn][k].second : seqNum[gn] - seqIdx[gn][k].second;
                int32_t refStart = seqIdx[gn][k].first;
                int32_t qryStart = seqIdx[gn][k].second;
                int32_t nIdx = k + gn*numBlocks;
                if (hostAlnLen[gn][k] <= 0) {
                    int32_t refLen = util->seqsLen[nodes[nIdx].first->identifier];
                    int32_t qryLen = util->seqsLen[nodes[nIdx].second->identifier];
                    std::vector<int8_t> aln;
                    alignGrpToGrp_traditional
                    (
                        freq[gn][k],
                        seqLen,
                        refLen,
                        qryLen,
                        param,
                        aln
                    );
                    int32_t alnLen = aln.size();
                    util->memCheck(alnLen);
                    std::reverse(aln.begin(), aln.end());
                    for (auto sIdx: tree->allNodes[nodes[nIdx].first->identifier]->msaIdx) {
                        int storeFrom = util->seqsStorage[sIdx];
                        int storeTo = 1 - util->seqsStorage[sIdx];
                        int orgIdx = 0;
                        for (int j = 0; j < aln.size(); ++j) {
                            if ((aln[j] & 0xFFFF) == 0 || (aln[j] & 0xFFFF) == 2) {
                                util->seqBuf[storeTo][sIdx][j] = util->seqBuf[storeFrom][sIdx][orgIdx];
                                orgIdx++;
                            }
                            else {
                                util->seqBuf[storeTo][sIdx][j] = '-';
                            }
                        }
                        util->seqsLen[nodes[nIdx].first->identifier] = aln.size();
                        util->changeStorage(sIdx);
                    }
                    for (auto sIdx: tree->allNodes[nodes[nIdx].second->identifier]->msaIdx) {
                        int storeFrom = util->seqsStorage[sIdx];
                        int storeTo = 1 - util->seqsStorage[sIdx];
                        int orgIdx = 0;
                        for (int j = 0; j < aln.size(); ++j) {
                            if ((aln[j] & 0xFFFF) == 0 || (aln[j] & 0xFFFF) == 1) {
                                util->seqBuf[storeTo][sIdx][j] = util->seqBuf[storeFrom][sIdx][orgIdx];
                                orgIdx++;
                            }
                            else {
                                util->seqBuf[storeTo][sIdx][j] = '-';
                            }
                        }
                        util->seqsLen[nodes[nIdx].second->identifier] = aln.size();
                        util->changeStorage(sIdx);
                    }
                    printf("CPU fallback (traditional global alignment) on No. %d (%s), Alignment Length: %d\n", k, tree->allNodes[nodes[nIdx].first->identifier]->identifier.c_str(), aln.size());
                    // printf("CPU fallback on No. %d (%s), Alignment Length: %d\n", k, tree->allNodes[nodes[nIdx].first->identifier]->identifier.c_str(), aln.size());
                }
                // else if (hostAlnLen[gn][k] <= 0) {
                //     std::vector<int8_t> aln;
                //     std::vector<std::vector<int>> freqRef;
                //     std::vector<std::vector<int>> freqQry;
                //     int32_t refLen = util->seqsLen[nodes[nIdx].first->identifier];
                //     int32_t qryLen = util->seqsLen[nodes[nIdx].second->identifier];
                //     for (int r = 0; r < refLen; r++) {
                //         std::vector<int> temp;
                //         for (int f = 0; f < 6; ++f) temp.push_back(freq[gn][k][6*r+f]);
                //         freqRef.push_back(temp);
                //     }
                //     for (int q = 0; q < qryLen; q++) {
                //         std::vector<int> temp;
                //         for (int f = 0; f < 6; ++f) temp.push_back(freq[gn][k][6*(seqLen+q)+f]);
                //         freqQry.push_back(temp);
                //     }
                //     Talco_xdrop::Params talco_params(param.match, param.mismatch, param.gapOpen, param.gapExtend, 1000, param.marker);
                //     Talco_xdrop::Align_freq (
                //         talco_params,
                //         freqRef,
                //         freqQry,
                //         aln
                //     );
                //     util->memCheck(aln.size());
                //     for (auto sIdx: tree->allNodes[nodes[nIdx].first->identifier]->msaIdx) {
                //         int64_t start = sIdx*util->memLen;
                //         int storeFrom = util->seqsStorage[sIdx];
                //         int storeTo = 1 - util->seqsStorage[sIdx];
                //         int orgIdx = 0;
                //         for (int j = 0; j < aln.size(); ++j) {
                //             if ((aln[j] & 0xFFFF) == 0 || (aln[j] & 0xFFFF) == 2) {
                //                 util->seqBuf[storeTo][start+j] = util->seqBuf[storeFrom][start+orgIdx];
                //                 orgIdx++;
                //             }
                //             else {
                //                 util->seqBuf[storeTo][start+j] = '-';
                //             }
                //         }
                //         util->seqsLen[nodes[nIdx].first->identifier] = aln.size();
                //         util->changeStorage(sIdx);
                //     }
                //     for (auto sIdx: tree->allNodes[nodes[nIdx].second->identifier]->msaIdx) {
                //         int64_t start = sIdx*util->memLen;
                //         int storeFrom = util->seqsStorage[sIdx];
                //         int storeTo = 1 - util->seqsStorage[sIdx];
                //         int orgIdx = 0;
                //         for (int j = 0; j < aln.size(); ++j) {
                //             if ((aln[j] & 0xFFFF) == 0 || (aln[j] & 0xFFFF) == 1) {
                //                 util->seqBuf[storeTo][start+j] = util->seqBuf[storeFrom][start+orgIdx];
                //                 orgIdx++;
                //             }
                //             else {
                //                 util->seqBuf[storeTo][start+j] = '-';
                //             }
                //         }
                //         util->seqsLen[nodes[nIdx].second->identifier] = aln.size();
                //         util->changeStorage(sIdx);
                //     }
                //     printf("CPU fallback (TALCO-Xdrop) on No. %d (%s), Alignment Length: %d\n", k, tree->allNodes[nodes[nIdx].first->identifier]->identifier.c_str(), aln.size());
                // }
                else {
                    for (auto sIdx: tree->allNodes[nodes[nIdx].first->identifier]->msaIdx) {
                        int orgIdx = 0;
                        int storeFrom = util->seqsStorage[sIdx];
                        int storeTo = 1 - util->seqsStorage[sIdx];
                        for (int j = 0; j < hostAlnLen[gn][k]; ++j) {
                            if ((hostAln[gn][k*2*seqLen+j] & 0xFFFF) == 0 || (hostAln[gn][k*2*seqLen+j] & 0xFFFF) == 2) {
                                util->seqBuf[storeTo][sIdx][j] = util->seqBuf[storeFrom][sIdx][orgIdx];
                                orgIdx++;
                            }
                            else {
                                util->seqBuf[storeTo][sIdx][j] = '-';
                            }
                        }
                        util->seqsLen[nodes[nIdx].first->identifier] = hostAlnLen[gn][k];
                        util->changeStorage(sIdx);
                    }
                    for (auto sIdx: tree->allNodes[nodes[nIdx].second->identifier]->msaIdx) {
                        int storeFrom = util->seqsStorage[sIdx];
                        int storeTo = 1 - util->seqsStorage[sIdx];
                        int orgIdx = 0;
                        for (int j = 0; j < hostAlnLen[gn][k]; ++j) {
                            if ((hostAln[gn][k*2*seqLen+j] & 0xFFFF) == 0 || (hostAln[gn][k*2*seqLen+j] & 0xFFFF) == 1) {
                                util->seqBuf[storeTo][sIdx][j] = util->seqBuf[storeFrom][sIdx][orgIdx];
                                orgIdx++;
                            }
                            else {
                                util->seqBuf[storeTo][sIdx][j] = '-';
                            }
                        }
                        util->seqsLen[nodes[nIdx].second->identifier] = hostAlnLen[gn][k];
                        util->changeStorage(sIdx);
                    }
                }
                // std::cout << "LenB : " << nodes[nIdx].first->identifier << '(' << tree->allNodes[nodes[nIdx].first->identifier]->msaIdx.size() << ')'
                //                       << nodes[nIdx].second->identifier << '(' << tree->allNodes[nodes[nIdx].second->identifier]->msaIdx.size() << ")\n";
                for (auto q: tree->allNodes[nodes[nIdx].second->identifier]->msaIdx) 
                    tree->allNodes[nodes[nIdx].first->identifier]->msaIdx.push_back(q);
                for (auto q: tree->allNodes[nodes[nIdx].second->identifier]->msa) 
                    tree->allNodes[nodes[nIdx].first->identifier]->msa.push_back(q);
            }  
        });
        for (int i = 0; i < alignSize[gn]; ++i) delete [] freq[gn][i];
    } 
    auto reAlnEnd = std::chrono::high_resolution_clock::now();
    std::chrono::nanoseconds reAlnTime = reAlnEnd - kernelEnd;
    printf("Alignment Time: %d us\n", reAlnTime.count() / 1000);

    

    for (int rn = 0; rn < roundGPU; ++rn) {
        free(hostFreq[rn]);
        free(hostAlnLen[rn]);
        free(hostLen[rn]);
        free(hostAln[rn]);
        free(hostSeqInfo[rn]);
    }  
    free(hostParam);

    delete [] alignSize;
    delete [] seqNum;
    delete [] deviceFreq;
    delete [] deviceAlnLen;
    delete [] deviceAln;
    delete [] deviceParam;
    delete [] deviceSeqInfo;
    delete [] hostFreq;
    delete [] hostAlnLen;
    delete [] hostAln;
    // delete [] hostParam;
    delete [] hostSeqInfo;
    return;
}

void createOverlapMSA_subtree(Tree* tree, std::vector<std::pair<Node*, Node*>> nodes, msa::utility* util, Params& param)
{

    int numBlocks = 1024; 
    int blockSize = THREAD_NUM;
    int gpuNum = util->gpuNum;
    // hipGetDeviceCount(&gpuNum); // number of CUDA devices
    
    // get maximum sequence/profile length 
    int32_t seqLen = util->memLen;
    int roundGPU = nodes.size() / numBlocks + 1;
    if (nodes.size()%numBlocks == 0) roundGPU -= 1;
    if (roundGPU < gpuNum) gpuNum = roundGPU;

    paramType* hostParam = (paramType*)malloc(28 * sizeof(paramType)); 

    if (!param.userDefine) {
        for (int i = 0; i < 5; ++i) {
            for (int j = 0; j < 5; ++j) {
                if (i == 5 || j == 5)          hostParam[i*5+j] = 0;
                else if (i == j)               hostParam[i*5+j] = param.match;
                else if (i-j == 2 || j-i == 2) hostParam[i*5+j] = param.trans;
                else                           hostParam[i*5+j] = param.mismatch;
            }
        }
        hostParam[25] = param.gapOpen;
        hostParam[26] = param.gapExtend;
        hostParam[27] = param.xdrop;
    }
    else {
        for (int i = 0; i < 5; ++i) for (int j = 0; j < 5; ++j) hostParam[i*5+j] = param.userMatrix[i][j];
        hostParam[25] = param.userGapOpen;
        hostParam[26] = param.userGapExtend;
        hostParam[27] = param.xdrop;
    }

    std::vector<std::vector<std::pair<int32_t, int32_t>>> seqIdx;
    
    uint16_t** hostFreq = new uint16_t* [gpuNum];
    int8_t**   hostAln = new int8_t* [gpuNum];
    int32_t**  hostLen = new int32_t* [gpuNum];
    int32_t**  hostAlnLen = new int32_t* [gpuNum];
    int32_t**  hostSeqInfo = new int32_t* [gpuNum];

    uint16_t** deviceFreq = new uint16_t* [gpuNum];
    int8_t**   deviceAln = new int8_t* [gpuNum];
    int32_t**  deviceLen = new int32_t* [gpuNum];
    int32_t**  deviceAlnLen = new int32_t* [gpuNum];
    int32_t**  deviceSeqInfo = new int32_t* [gpuNum];
    paramType**  deviceParam = new paramType* [gpuNum];
  
    std::atomic<int> nowRound;
    nowRound.store(0);

    tbb::parallel_for(tbb::blocked_range<int>(0, gpuNum), [&](tbb::blocked_range<int> range){ 
        for (int gn = range.begin(); gn < range.end(); ++gn) {
            hostFreq[gn] = (uint16_t*)malloc(12 * seqLen * numBlocks * sizeof(uint16_t));
            hostAln[gn] = (int8_t*)malloc(    2 * seqLen * numBlocks * sizeof(int8_t));
            hostLen[gn] = (int32_t*)malloc(   2 *          numBlocks * sizeof(int32_t));
            hostAlnLen[gn] = (int32_t*)malloc(             numBlocks * sizeof(int32_t));
            hostSeqInfo[gn] = (int32_t*)malloc(5 * sizeof(int32_t));
            
            hipSetDevice(gn);
            // hipError_t error;
            hipMalloc((void**)&deviceFreq[gn],  12 * seqLen * numBlocks * sizeof(uint16_t));
            hipMalloc((void**)&deviceAln[gn],    2 * seqLen * numBlocks * sizeof(int8_t));
            hipMalloc((void**)&deviceLen[gn],    2 *          numBlocks * sizeof(int32_t));
            hipMalloc((void**)&deviceAlnLen[gn],              numBlocks * sizeof(int32_t));
            hipMalloc((void**)&deviceSeqInfo[gn], 5 * sizeof(int32_t));
            hipMalloc((void**)&deviceParam[gn],  28 * sizeof(paramType));

            hipMemcpy(deviceParam[gn], hostParam, 28 * sizeof(paramType), hipMemcpyHostToDevice);
            // error = hipGetLastError(); printf("CUDA error Malloc: %s\n",hipGetErrorString(error)); 
            std::vector<std::pair<int, int>> seqIdx;
            
            while (nowRound < roundGPU) {
                int rn = nowRound.fetch_add(1);
                int alnPairs = (nodes.size() - rn*numBlocks > numBlocks) ? numBlocks : nodes.size() - rn*numBlocks;
                int seqNum = 0;
                // std::cout << "GPU: " << gn << " Rn: " << rn << " Pairs: " << alnPairs << '\n';
                
                // Initailize 
                for (int n = 0; n < 12*seqLen * numBlocks; ++n) hostFreq[gn][n] = 0;
                for (int n = 0; n <  2*seqLen * numBlocks; ++n) hostAln[gn][n] = 0;
                for (int n = 0; n <  2*         numBlocks; ++n) hostLen[gn][n] = 0;
                for (int n = 0; n <             numBlocks; ++n) hostAlnLen[gn][n] = 0;
                seqIdx.clear();

                // Calculate Frequency
                for (int n = 0; n < alnPairs; ++n) {
                    int32_t nIdx = n + rn*numBlocks;
                    int32_t qryIdx = 0;
                    int32_t refIdx = 0;
                    int32_t refLen = util->seqsLen[nodes[nIdx].first->identifier];
                    int32_t qryLen = util->seqsLen[nodes[nIdx].second->identifier];
                    // std::cout << n << "Len: " << refLen << ',' << qryLen << '\n';
                    refIdx = seqNum;
                    for (auto sIdx: tree->allNodes[nodes[nIdx].first->identifier]->msaIdx) { 
                        int storage = util->seqsStorage[sIdx];
                        tbb::parallel_for(tbb::blocked_range<int>(0, refLen), [&](tbb::blocked_range<int> r) {
                        for (int s = r.begin(); s < r.end(); ++s) {
                        // for (int s = 0; s < refLen; ++s) {
                            if      (util->alnStorage[storage][sIdx][s] == 'A' || util->alnStorage[storage][sIdx][s] == 'a') hostFreq[gn][12*seqLen*n+6*s+0]+=1;
                            else if (util->alnStorage[storage][sIdx][s] == 'C' || util->alnStorage[storage][sIdx][s] == 'c') hostFreq[gn][12*seqLen*n+6*s+1]+=1;
                            else if (util->alnStorage[storage][sIdx][s] == 'G' || util->alnStorage[storage][sIdx][s] == 'g') hostFreq[gn][12*seqLen*n+6*s+2]+=1;
                            else if (util->alnStorage[storage][sIdx][s] == 'T' || util->alnStorage[storage][sIdx][s] == 't' ||
                                     util->alnStorage[storage][sIdx][s] == 'U' || util->alnStorage[storage][sIdx][s] == 'u') hostFreq[gn][12*seqLen*n+6*s+3]+=1;
                            else if (util->alnStorage[storage][sIdx][s] == 'N' || util->alnStorage[storage][sIdx][s] == 'n') hostFreq[gn][12*seqLen*n+6*s+4]+=1;
                            else                                                                                             hostFreq[gn][12*seqLen*n+6*s+5]+=1;
                        }
                        });
                        seqNum += 1;
                    }
                    qryIdx = seqNum;
                    for (auto sIdx: tree->allNodes[nodes[nIdx].second->identifier]->msaIdx) { 
                        int storage = util->seqsStorage[sIdx];
                        tbb::parallel_for(tbb::blocked_range<int>(0, qryLen), [&](tbb::blocked_range<int> r) {
                        for (int s = r.begin(); s < r.end(); ++s) {
                        // for (int s = 0; s < qryLen; ++s) {
                            if      (util->alnStorage[storage][sIdx][s] == 'A' || util->alnStorage[storage][sIdx][s] == 'a') hostFreq[gn][12*seqLen*n+6*(seqLen+s)+0]+=1;
                            else if (util->alnStorage[storage][sIdx][s] == 'C' || util->alnStorage[storage][sIdx][s] == 'c') hostFreq[gn][12*seqLen*n+6*(seqLen+s)+1]+=1;
                            else if (util->alnStorage[storage][sIdx][s] == 'G' || util->alnStorage[storage][sIdx][s] == 'g') hostFreq[gn][12*seqLen*n+6*(seqLen+s)+2]+=1;
                            else if (util->alnStorage[storage][sIdx][s] == 'T' || util->alnStorage[storage][sIdx][s] == 't' ||
                                     util->alnStorage[storage][sIdx][s] == 'U' || util->alnStorage[storage][sIdx][s] == 'u') hostFreq[gn][12*seqLen*n+6*(seqLen+s)+3]+=1;
                            else if (util->alnStorage[storage][sIdx][s] == 'N' || util->alnStorage[storage][sIdx][s] == 'n') hostFreq[gn][12*seqLen*n+6*(seqLen+s)+4]+=1;
                            else                                                                                             hostFreq[gn][12*seqLen*n+6*(seqLen+s)+5]+=1;
                        }
                        });
                        seqNum += 1;
                    }
                    hostLen[gn][2*n] = refLen; hostLen[gn][2*n+1] = qryLen;
                    seqIdx.push_back(std::make_pair(refIdx, qryIdx));
                }

                hostSeqInfo[gn][0] = seqLen;
                hostSeqInfo[gn][1] = seqNum;
                hostSeqInfo[gn][2] = alnPairs;
                hostSeqInfo[gn][3] = numBlocks;
                hostSeqInfo[gn][4] = param.userDefine;
        
                hipMemcpy(deviceFreq[gn], hostFreq[gn], 12*seqLen * numBlocks * sizeof(uint16_t), hipMemcpyHostToDevice);
                hipMemcpy(deviceAln[gn], hostAln[gn], 2*seqLen * numBlocks * sizeof(int8_t), hipMemcpyHostToDevice);
                hipMemcpy(deviceLen[gn], hostLen[gn], 2*numBlocks * sizeof(int32_t), hipMemcpyHostToDevice);
                hipMemcpy(deviceAlnLen[gn], hostAlnLen[gn], numBlocks * sizeof(int32_t), hipMemcpyHostToDevice);
                hipMemcpy(deviceSeqInfo[gn], hostSeqInfo[gn], 5 * sizeof(int32_t), hipMemcpyHostToDevice);
                
                std::string berr = hipGetErrorString(hipGetLastError());
                if (berr != "no error") printf("ERROR: Before kernel %s!\n", berr.c_str());
                alignGrpToGrp_talco<<<numBlocks, blockSize>>>(
                    deviceFreq[gn],
                    deviceAln[gn], 
                    deviceLen[gn],
                    deviceAlnLen[gn],
                    deviceSeqInfo[gn], 
                    deviceParam[gn]
                );
                hipDeviceSynchronize();
                std::string aerr = hipGetErrorString(hipGetLastError());
                if (aerr != "no error") printf("ERROR: After kernel %s!\n", aerr.c_str());
                
                hipMemcpy(hostAln[gn], deviceAln[gn], 2*seqLen * numBlocks * sizeof(int8_t), hipMemcpyDeviceToHost);
                hipMemcpy(hostAlnLen[gn], deviceAlnLen[gn], numBlocks * sizeof(int32_t), hipMemcpyDeviceToHost);
                hipDeviceSynchronize();
                // int maxAlnLen = 0;
                // for (int n = 0; n <  alnPairs; ++n) {
                //     if (hostAlnLen[gn][n] > maxAlnLen) maxAlnLen = hostAlnLen[gn][n];
                // }
                // util->memCheck(maxAlnLen);
                
                
                // tbb::parallel_for(tbb::blocked_range<int>(0, alignSize[gn]), [&](tbb::blocked_range<int> range) {
                // for (int k = range.begin(); k < range.end(); ++k) {
                for (int n = 0; n < alnPairs; ++n) {
                    int32_t refNum = seqIdx[n].second - seqIdx[n].first;
                    int32_t qryNum = (n !=  alnPairs-1) ? seqIdx[n+1].first - seqIdx[n].second : seqNum - seqIdx[n].second;
                    int32_t nIdx = n + rn*numBlocks;

                    if (hostAlnLen[gn][n] <= 0) {
                        int32_t refLen = util->seqsLen[nodes[nIdx].first->identifier];
                        int32_t qryLen = util->seqsLen[nodes[nIdx].second->identifier];
                        uint16_t *freq = new uint16_t[12*seqLen]; 
                        for (int i = 0; i < 12*seqLen; ++i) freq[i] = hostFreq[gn][12*seqLen*n+i];
                        std::vector<int8_t> aln;
                        alignGrpToGrp_traditional (
                            freq,
                            seqLen,
                            refLen,
                            qryLen,
                            param,
                            aln
                        );
                        delete [] freq;
                        int32_t alnLen = aln.size();
                        util->memCheck(alnLen);
                        std::reverse(aln.begin(), aln.end());
                        tree->allNodes[nodes[nIdx].first->identifier]->msaAln = aln;
                        std::cout << "CPU fallback (traditional global alignment) on No. " << n << " (" << tree->allNodes[nodes[nIdx].first->identifier]->identifier << ")\n";
                    }
                    else {
                        std::vector<int8_t> aln;
                        for (int j = 0; j < hostAlnLen[gn][n]; ++j) {
                            aln.push_back(hostAln[gn][n*2*seqLen+j]);
                        }
                        tree->allNodes[nodes[nIdx].first->identifier]->msaAln = aln;
                    }
                }    
            }  
        }
    });
    
    for (auto n: nodes) {
        tree->allNodes[n.first->identifier]->msa.clear();
        tree->allNodes[n.first->identifier]->msa.push_back(n.first->identifier);
        tree->allNodes[n.second->identifier]->msa.clear();
        tree->allNodes[n.second->identifier]->msa.push_back(n.second->identifier);    
    }

    // free memory  
    for (int gn = 0; gn < gpuNum; ++gn) {
        hipSetDevice(gn);
        hipFree(deviceFreq[gn]);
        hipFree(deviceAlnLen[gn]);
        hipFree(deviceLen[gn]);
        hipFree(deviceAln[gn]);
        hipFree(deviceSeqInfo[gn]);
        hipFree(deviceParam[gn]);
        hipDeviceSynchronize();  
        free(hostFreq[gn]);
        free(hostAlnLen[gn]);
        free(hostLen[gn]);
        free(hostAln[gn]);
        free(hostSeqInfo[gn]);
    }
    
    free(hostParam);

    delete [] deviceFreq;
    delete [] deviceAlnLen;
    delete [] deviceAln;
    delete [] deviceParam;
    delete [] deviceSeqInfo;
    delete [] hostFreq;
    delete [] hostAlnLen;
    delete [] hostLen;
    delete [] hostAln;
    delete [] hostSeqInfo;
    return;
}

*/
